// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file bc_app.cu
 *
 * @brief Gunrock betweeness centrality (BC) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// betweeness centrality includes
#include <gunrock/app/bc/bc_enactor.cuh>
#include <gunrock/app/bc/bc_problem.cuh>
#include <gunrock/app/bc/bc_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bc;

/**
 * @brief Run betweenness centrality tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[out] graph_o Pointer to the output CSR graph object
 * @param[in] csr Reference to the CSR graph object defined in main driver
 * @param[in] source
 * @param[in] max_grid_size
 * @param[in] num_gpus
 * @param[in] max_queue_sizing
 * @param[in] context Reference to CudaContext used by moderngpu functions
 */
template<typename VertexId, typename Value, typename SizeT>
void run_bc(
    GRGraph        *graph_o,
    const Csr<VertexId, Value, SizeT> &csr,
    const VertexId source,
    const int      max_grid_size,
    const int      num_gpus,
    const double   max_queue_sizing,
    CudaContext    &context) {
    typedef BCProblem<VertexId, SizeT, Value, true, false > Problem;
    // Allocate host-side array (for both reference and gpu-computed results)
    Value *h_sigmas     = (Value*)malloc(sizeof(Value) * csr.nodes);
    Value *h_bc_values  = (Value*)malloc(sizeof(Value) * csr.nodes);
    Value *h_ebc_values = (Value*)malloc(sizeof(Value) * csr.edges);
    BCEnactor<false> enactor(false);  // Allocate BC enactor map
    Problem *problem = new Problem;   // Allocate problem on GPU

    util::GRError(problem->Init(false, csr, num_gpus),
                  "BC Problem Initialization Failed", __FILE__, __LINE__);

    VertexId start_source;
    VertexId end_source;
    if (source == -1) {
        start_source = 0;
        end_source = csr.nodes;
    } else {
        start_source = source;
        end_source = source + 1;
    }

    for (VertexId i = start_source; i < end_source; ++i) {
        util::GRError(problem->Reset(
                          i, enactor.GetFrontierType(), max_queue_sizing),
                      "BC Problem Data Reset Failed", __FILE__, __LINE__);
        util::GRError(enactor.template Enact<Problem>(
                          context, problem, i, max_grid_size),
                      "BC Problem Enact Failed", __FILE__, __LINE__);
    }

    util::MemsetScaleKernel <<< 128, 128>>>(
        problem->data_slices[0]->d_bc_values, (Value)0.5f, (int)csr.nodes);

    util::GRError(problem->Extract(h_sigmas, h_bc_values, h_ebc_values),
                  "BC Problem Data Extraction Failed", __FILE__, __LINE__);

    graph_o->node_values = (float*)&h_bc_values[0];   // h_bc_values per node 
    graph_o->edge_values = (float*)&h_ebc_values[0];  // h_ebc_values per edge

    if (problem) { delete problem; }
    hipDeviceSynchronize();
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] graph_o  GRGraph type output
 * @param[in]  graph_i  GRGraph type input graph
 * @param[in]  config   Specific configurations
 * @param[in]  data_t   Data type configurations
 * @param[in]  context  ModernGPU context
 */
void dispatch_bc(
    GRGraph       *graph_o,
    const GRGraph *graph_i,
    const GRSetup  config,
    const GRTypes  data_t,
    CudaContext   &context) {
    switch (data_t.VTXID_TYPE) {
    case VTXID_INT: {
        switch (data_t.SIZET_TYPE) {
        case SIZET_INT: {
            switch (data_t.VALUE_TYPE) {
            case VALUE_INT: {  // template type = <int, int, int>
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_UINT: {  // template type = <int, uint, int>
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT: {  // template type = <int, float, int>
                // build input csr format graph
                Csr<int, float, int> csr_graph(false);
                csr_graph.nodes = graph_i->num_nodes;
                csr_graph.edges = graph_i->num_edges;
                csr_graph.row_offsets    = (int*)graph_i->row_offsets;
                csr_graph.column_indices = (int*)graph_i->col_indices;

                // bc configurations
                int   src_node         =  -1;  // default source vertex to start
                int   max_grid_size    =   0;  // leave it up to the enactor
                int   num_gpus         =   1;  // Number of GPUs for multi-gpu
                float max_queue_sizing = 1.0;  // Maximum size scaling factor

                // determine source vertex to start bc
                switch (config.src_mode) {
                case randomize: {
                    src_node = graphio::RandomNode(csr_graph.nodes);
                    break;
                }
                case largest_degree: {
                    int max_deg = 0;
                    src_node = csr_graph.GetNodeWithHighestDegree(max_deg);
                    break;
                }
                case manually: {
                    src_node = config.src_node;
                    break;
                }
                default: {
                    src_node = 0;
                    break;
                }
                }
                max_queue_sizing = config.queue_size;

                // lunch bc function
                run_bc<int, float, int>(
                    graph_o,
                    csr_graph,
                    src_node,
                    max_grid_size,
                    num_gpus,
                    max_queue_sizing,
                    context);

                // reset for free memory
                csr_graph.row_offsets    = NULL;
                csr_graph.column_indices = NULL;
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/*
 * @brief gunrock_bc function
 *
 * @param[out] graph_o output of bc problem
 * @param[in]  graph_i input graph need to process on
 * @param[in]  config  gunrock primitive specific configurations
 * @param[in]  data_t  gunrock data_t struct
 */
void gunrock_bc(
    GRGraph       *graph_o,
    const GRGraph *graph_i,
    const GRSetup  config,
    const GRTypes  data_t) {
    unsigned int device = 0;
    device = config.device;
    ContextPtr context = mgpu::CreateCudaDevice(device);
    dispatch_bc(graph_o, graph_i, config, data_t, *context);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
