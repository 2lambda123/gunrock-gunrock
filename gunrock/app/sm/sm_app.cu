// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file ss_app.cu
 *
 * @brief subgraph matching (SM) application
 */

#include <gunrock/app/sm/sm_app.cuh>

namespace gunrock {
namespace app {
namespace sm {

template <typename ParametersT>
hipError_t UseParameters(ParametersT &parameters) {
  hipError_t retval = hipSuccess;
  GUARD_CU(UseParameters_app(parameters));
  GUARD_CU(UseParameters_problem(parameters));
  GUARD_CU(UseParameters_enactor(parameters));
  GUARD_CU(UseParameters_test(parameters));

  GUARD_CU(parameters.Use<unsigned int>(
              "num-subgraphs",
              util::REQUIRED_ARGUMENT | util::SINGLE_VALUE | util::INTERNAL_PARAMETER,
              0, "number of matched subgraphs", __FILE__, __LINE__));

  return retval;
}

} // namespace sm
} // namespace app
} // namespace gunrock

/*
 * @brief Simple interface take in graph as CSR format
 * @param[in]  num_nodes   Number of veritces in the input graph
 * @param[in]  num_edges   Number of edges in the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  edge_values CSR-formatted graph input edge weights
 * @param[in]  num_runs    Number of runs to perform SM
 * @param[out] subgraphs   Return number of subgraphs
 * \return     double      Return accumulated elapsed times for all runs
 */
double sm(
    const int            num_nodes,
    const int            num_edges,
    const int           *row_offsets,
    const int           *col_indices,
    const unsigned long *edge_values,
    const int            num_runs,
          int           *subgraphs)
{
    return sm(num_nodes, num_edges, row_offsets, col_indices,
        edge_values, 1 /* num_runs */, subgraphs);
}

/*
 * @brief Simple interface take in graph as Gunrock format
 * @param[in]  query_graph Query graph to be searched
 * @param[in]  data_graph  data graph to be searched on
 * @param[in]  num_runs    Number of runs to perform SM
 * @param[out] subgraphs   Return number of subgraphs
 * \return     double      Return accumulated elapsed times for all runs
 */
double nv_sm(
    gunrock::app::TestGraph<int, int, unsigned long,
    gunrock::graph::HAS_CSR> &query_graph,
    gunrock::app::TestGraph<int, int, unsigned long,
    gunrock::graph::HAS_CSR> &data_graph,
    const int            num_runs,
          int           *subgraphs)
{
    return nv_sm(query_graph, data_graph, 1 /* num_runs */, subgraphs);
}
// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
