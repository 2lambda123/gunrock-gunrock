// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file sssp_app.cu
 *
 * @brief single-source shortest path (SSSP) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// single-source shortest path includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;

struct Test_Parameter : gunrock::app::TestParameter_Base {
  public:
    bool   mark_predecessors;
    int    delta_factor;
    double max_queue_sizing1;

    Test_Parameter() {
        delta_factor      =    32;
        mark_predecessors = false;
        max_queue_sizing1 =  -1.0;
    }

    ~Test_Parameter() {
    }
};

template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS >
void runSSSP(GRGraph* output, Test_Parameter *parameter);

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT,
    bool        DEBUG,
    bool        SIZE_CHECK >
void markPredecessorsSSSP(GRGraph* output, Test_Parameter *parameter) {
    if (parameter->mark_predecessors)
        runSSSP<VertexId, Value, SizeT, INSTRUMENT,
                DEBUG, SIZE_CHECK,  true>(output, parameter);
    else
        runSSSP<VertexId, Value, SizeT, INSTRUMENT,
                DEBUG, SIZE_CHECK, false>(output, parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void sizeCheckSSSP(GRGraph* output, Test_Parameter *parameter) {
    if (parameter->size_check)
        markPredecessorsSSSP<VertexId, Value, SizeT, INSTRUMENT,
                             DEBUG,  true>(output, parameter);
    else
        markPredecessorsSSSP<VertexId, Value, SizeT, INSTRUMENT,
                             DEBUG, false>(output, parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void debugSSSP(GRGraph* output, Test_Parameter *parameter) {
    if (parameter->debug)
        sizeCheckSSSP<VertexId, Value, SizeT, INSTRUMENT,
                      true>(output, parameter);
    else
        sizeCheckSSSP<VertexId, Value, SizeT, INSTRUMENT,
                      false>(output, parameter);
}

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT >
void instrumentedSSSP(GRGraph* output, Test_Parameter *parameter) {
    if (parameter->instrumented)
        debugSSSP<VertexId, Value, SizeT,  true>(output, parameter);
    else
        debugSSSP<VertexId, Value, SizeT, false>(output, parameter);
}

/**
 * @brief Run SSSP tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 * @tparam MARK_PREDECESSORS
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node where SSSP starts
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] queue_sizing Scaling factor used in edge mapping
 * @param[in] num_gpus Number of GPUs
 * @param[in] delta_factor Parameter to specify delta in delta-stepping SSSP
 * @param[in] iterations Number of iteration for running the test
 & @param[in] traversal_mode Load-balanced or Dynamic cooperative
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK,
    bool MARK_PREDECESSORS >
void runSSSP(GRGraph* output, Test_Parameter *parameter) {
    typedef SSSPProblem < VertexId,
            SizeT,
            Value,
            MARK_PREDECESSORS > SsspProblem;

    typedef SSSPEnactor < SsspProblem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK > SsspEnactor;

    Csr<VertexId, Value, SizeT>
    *graph = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    VertexId      src                = (VertexId)parameter -> src;
    int           max_grid_size      = parameter -> max_grid_size;
    int           num_gpus           = parameter -> num_gpus;
    double        max_queue_sizing   = parameter -> max_queue_sizing;
    double        max_in_sizing      = parameter -> max_in_sizing;
    ContextPtr   *context            = (ContextPtr*)parameter -> context;
    std::string   partition_method   = parameter -> partition_method;
    int          *gpu_idx            = parameter -> gpu_idx;
    hipStream_t *streams            = parameter -> streams;
    float         partition_factor   = parameter -> partition_factor;
    int           partition_seed     = parameter -> partition_seed;
    bool          g_stream_from_host = parameter -> g_stream_from_host;
    int           delta_factor       = parameter -> delta_factor;
    int           traversal_mode     = parameter -> traversal_mode;
    size_t       *org_size           = new size_t[num_gpus];
    // Allocate host-side label arrays
    Value    *h_labels = new Value[graph->nodes];
    VertexId *h_preds  = MARK_PREDECESSORS ? new VertexId[graph->nodes] : NULL;

    for (int gpu = 0; gpu < num_gpus; gpu++) {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    SsspEnactor* enactor = new SsspEnactor(num_gpus, gpu_idx);  // enactor map
    SsspProblem* problem = new SsspProblem;  // Allocate problem on GPU

    util::GRError(
        problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            delta_factor,
            max_queue_sizing,
            max_in_sizing,
            partition_factor,
            partition_seed),
        "Problem SSSP Initialization Failed", __FILE__, __LINE__);

    util::GRError(
        enactor->Init (context, problem, max_grid_size, traversal_mode),
        "SSSP Enactor init failed", __FILE__, __LINE__);

    // Perform SSSP
    CpuTimer cpu_timer;

    util::GRError(
        problem->Reset(src, enactor->GetFrontierType(), max_queue_sizing),
        "SSSP Problem Data Reset Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Reset(), "SSSP Enactor Reset failed", __FILE__, __LINE__);

    printf("__________________________\n"); fflush(stdout);
    cpu_timer.Start();
    util::GRError(
        enactor->Enact(src, traversal_mode),
        "SSSP Problem Enact Failed", __FILE__, __LINE__);
    cpu_timer.Stop();
    printf("--------------------------\n"); fflush(stdout);
    float elapsed = cpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(
        problem->Extract(h_labels, h_preds),
        "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

    output->node_value1 = (Value*)&h_labels[0];
    if (MARK_PREDECESSORS) output->node_value2 = (VertexId*)&h_preds[0];

    printf(" GPU Single-Source Shortest Path finished in %lf msec.\n", elapsed);

    // Clean up
    if (org_size) { delete[] org_size; org_size = NULL; }
    if (enactor ) { delete   enactor ; enactor  = NULL; }
    if (problem ) { delete   problem ; problem  = NULL; }
    // if (h_labels) { delete[] h_labels; h_labels = NULL; }
    // if (h_preds ) { delete[] h_preds ; h_preds  = NULL; }
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] graph_o     GRGraph type output
 * @param[out] predecessor Return predeessor if mark_pred = true
 * @param[in]  graph_i     GRGraph type input graph
 * @param[in]  config      Primitive-specific configurations
 * @param[in]  data_t      Data type configurations
 * @param[in]  context     ModernGPU context
 */
void dispatchSSSP(
    GRGraph*       graph_o,
    const GRGraph* graph_i,
    const GRSetup  config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams) {
    Test_Parameter *parameter = new Test_Parameter;
    parameter->context  = context;
    parameter->streams  = streams;
    parameter->num_gpus = config.num_devices;
    parameter->gpu_idx  = config.device_list;
    parameter->delta_factor = config.delta_factor;
    parameter->traversal_mode = config.traversal_mode;
    parameter->mark_predecessors  = config.mark_predecessors;

    switch (data_t.VTXID_TYPE) {
    case VTXID_INT: {
        switch (data_t.SIZET_TYPE) {
        case SIZET_INT: {
            switch (data_t.VALUE_TYPE) {
            case VALUE_INT: {  // template type = <int, int, int>
                Csr<int, int, int> csr(false);
                csr.nodes = graph_i->num_nodes;
                csr.edges = graph_i->num_edges;
                csr.row_offsets    = (int*)graph_i->row_offsets;
                csr.column_indices = (int*)graph_i->col_indices;
                csr.edge_values    = (int*)graph_i->edge_values;
                parameter->graph = &csr;

                // determine source vertex to start
                switch (config.source_mode) {
                case randomize: {
                    parameter->src = graphio::RandomNode(csr.nodes);
                    break;
                }
                case largest_degree: {
                    int max_deg = 0;
                    parameter->src = csr.GetNodeWithHighestDegree(max_deg);
                    break;
                }
                case manually: {
                    parameter->src = config.source_vertex;
                    break;
                }
                default: {
                    parameter->src = 0;
                    break;
                }
                }
                printf(" source: %lld\n", (long long) parameter->src);
                instrumentedSSSP<int, int, int>(graph_o, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                csr.edge_values    = NULL;
                break;
            }
            case VALUE_UINT: {  // template type = <int, uint, int>
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT: {
                // template type = <int, float, int>
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/**
 * @brief run_sssp entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[out] graph_o     GRGraph type output
 * @param[in]  graph_i     GRGraph type input graph
 * @param[in]  config      Primitive specific configurations
 * @param[in]  data_t      Data type configurations
 */
void gunrock_sssp(
    GRGraph*       graph_o,
    const GRGraph* graph_i,
    const GRSetup  config,
    const GRTypes  data_t) {
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config.num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i) {
        gpu_idx[i] = config.device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    printf(" using %d GPUs:", num_gpus);
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        printf(" %d ", gpu_idx[gpu]);
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i) {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus) {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    printf("\n");

    dispatchSSSP(graph_o, graph_i, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 * @param[out] distances   Return shortest distance to source per nodes
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  source      Source to begin traverse
 */
void sssp(
    unsigned int*       distances,
    const int           num_nodes,
    const int           num_edges,
    const int*          row_offsets,
    const int*          col_indices,
    const unsigned int* edge_values,
    const int           source) {
    struct GRTypes data_t;           // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;   // integer
    data_t.SIZET_TYPE = SIZET_INT;   // integer
    data_t.VALUE_TYPE = VALUE_INT;  // unsigned integer

    struct GRSetup config;                // primitive-specific configures
    int list[] = {0, 1, 2, 3};            // device to run algorithm
    config.num_devices = sizeof(list) / sizeof(list[0]);  // number of devices
    config.device_list       = list;      // device list to run algorithm
    config.source_mode       = manually;  // manually setting source vertex
    config.source_vertex     = source;    // source vertex to start
    config.delta_factor      =    32;     // delta factor for SSSP
    config.mark_predecessors = false;     // do not mark predecessors
    config.traversal_mode    =     0;     // 0 for Load balanced partition
    config.max_queue_sizing  =  1.0f;     // maximum queue sizing factor

    struct GRGraph *graph_o = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graph_i = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graph_i->num_nodes   = num_nodes;
    graph_i->num_edges   = num_edges;
    graph_i->row_offsets = (void*)&row_offsets[0];
    graph_i->col_indices = (void*)&col_indices[0];
    graph_i->edge_values = (void*)&edge_values[0];

    printf(" loaded %d nodes and %d edges\n", num_nodes, num_edges);

    gunrock_sssp(graph_o, graph_i, config, data_t);
    memcpy(distances, (int*)graph_o->node_value1, num_nodes * sizeof(int));

    if (graph_i) free(graph_i);
    if (graph_o) free(graph_o);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
