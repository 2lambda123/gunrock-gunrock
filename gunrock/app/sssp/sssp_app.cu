// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file sssp_app.cu
 *
 * @brief single-source shortest path (SSSP) application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// single-source shortest path includes
#include <gunrock/app/sssp/sssp_enactor.cuh>
#include <gunrock/app/sssp/sssp_problem.cuh>
#include <gunrock/app/sssp/sssp_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::sssp;

/**
 * @brief run single-source shortest path procedures
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam MARK_PREDECESSORS
 *
 * @param[out] graph_o GRGraph type output
 * @param[out] predecessor return predeessor if mark_pred = true
 * @param[in]  graph Reference to the CSR graph we process on
 * @param[in]  source Source node where SSSP starts
 * @param[in]  max_grid_size Maximum CTA occupancy
 * @param[in]  queue_sizing Scaling factor used in edge mapping
 * @param[in]  num_gpus Number of GPUs
 * @param[in]  delta_factor user set
 * @param[in]  context moderngpu context
 */
template<typename VertexId, typename Value, typename SizeT, 
         bool MARK_PREDECESSORS>
void run_sssp(
    GRGraph        *graph_o,
    VertexId       *predecessor,
    const Csr<VertexId, Value, SizeT> &csr,
    const VertexId src,
    const int      max_grid_size,
    const float    queue_sizing,
    const int      num_gpus,
    const int      delta_factor,
    CudaContext    &context) {
    typedef SSSPProblem<VertexId, SizeT, Value, MARK_PREDECESSORS> Problem;
    // Allocate host-side label array for gpu-computed results
    Value *h_labels = (Value*)malloc(sizeof(Value) * csr.nodes);
    //VertexId     *h_preds  = NULL;

    if (MARK_PREDECESSORS) {
        //h_preds = (VertexId*)malloc(sizeof(VertexId) * csr.nodes);
    }

    SSSPEnactor<false> enactor(false);  // enactor map
    Problem *problem = new Problem;
    util::GRError(problem->Init(false, csr, num_gpus, delta_factor),
                  "SSSP Problem Initialization Failed", __FILE__, __LINE__);

    util::GRError(problem->Reset(src, enactor.GetFrontierType(), queue_sizing),
                  "SSSP Problem Data Reset Failed", __FILE__, __LINE__);

    util::GRError(enactor.template Enact<Problem>(
                      context, problem, src, queue_sizing, max_grid_size),
                  "SSSP Problem Enact Failed", __FILE__, __LINE__);

    util::GRError(problem->Extract(h_labels, predecessor),
                  "SSSP Problem Data Extraction Failed", __FILE__, __LINE__);

    // copy label_values per node to GRGraph output
    graph_o->node_values = (Value*)&h_labels[0];

    if (problem) { delete problem; }
    hipDeviceSynchronize();
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] graph_o     GRGraph type output
 * @param[out] predecessor Return predeessor if mark_pred = true
 * @param[in]  graph_i     GRGraph type input graph
 * @param[in]  config      Primitive-specific configurations
 * @param[in]  data_t      Data type configurations
 * @param[in]  context     ModernGPU context
 */
void dispatch_sssp(
    GRGraph       *graph_o,
    void          *predecessor,
    const GRGraph *graph_i,
    const GRSetup config,
    const GRTypes data_t,
    CudaContext   &context) {
    switch (data_t.VTXID_TYPE) {
    case VTXID_INT: {
        switch (data_t.SIZET_TYPE) {
        case SIZET_INT: {
            switch (data_t.VALUE_TYPE) {
            case VALUE_INT: {  // template type = <int, int, int>
                Csr<int, int, int> csr_graph(false);
                csr_graph.nodes          = graph_i->num_nodes;
                csr_graph.edges          = graph_i->num_edges;
                csr_graph.row_offsets    = (int*)graph_i->row_offsets;
                csr_graph.column_indices = (int*)graph_i->col_indices;
                csr_graph.edge_values    = (int*)graph_i->edge_values;

                // sssp configurations
                bool  mark_pred        =   0;  // whether to mark predecessors
                int   src_node         =   0;  // source vertex to start
                int   num_gpus         =   1;  // number of GPUs
                int   delta_factor     =   1;  // default delta_factor = 1
                int   max_grid_size    =   0;  // leave it up to the enactor
                float max_queue_sizing = 1.0;  // default maximum queue sizing

                // determine source vertex to start sssp
                switch (config.src_mode) {
                case randomize: {
                    src_node = graphio::RandomNode(csr_graph.nodes);
                    break;
                }
                case largest_degree: {
                    int max_deg = 0;
                    src_node = csr_graph.GetNodeWithHighestDegree(max_deg);
                    break;
                }
                case manually: {
                    src_node = config.src_node;
                    break;
                }
                default: {
                    src_node = 0;
                    break;
                }
                }
                mark_pred        = config.mark_pred;
                delta_factor     = config.delta_factor;
                max_queue_sizing = config.queue_size;

                switch (mark_pred) {
                case true: {
                    run_sssp<int, int, int, true>(
                        graph_o,
                        (int*)predecessor,
                        csr_graph,
                        src_node,
                        max_grid_size,
                        max_queue_sizing,
                        num_gpus,
                        delta_factor,
                        context);
                    break;
                }
                case false: {
                    run_sssp<int, int, int, false>(
                        graph_o,
                        (int*)predecessor,
                        csr_graph,
                        src_node,
                        max_grid_size,
                        max_queue_sizing,
                        num_gpus,
                        delta_factor,
                        context);
                    break;
                }
                }
                // reset for free memory
                csr_graph.row_offsets    = NULL;
                csr_graph.column_indices = NULL;
                csr_graph.edge_values    = NULL;
                break;
            }
            case VALUE_UINT: {  // template type = <int, uint, int>
                // build input csr format graph
                Csr<int, unsigned int, int> csr_graph(false);
                csr_graph.nodes          = graph_i->num_nodes;
                csr_graph.edges          = graph_i->num_edges;
                csr_graph.row_offsets    = (int*)graph_i->row_offsets;
                csr_graph.column_indices = (int*)graph_i->col_indices;
                csr_graph.edge_values    = (unsigned int*)graph_i->edge_values;

                // sssp configurations
                bool  mark_pred        =   0;  // whether to mark predecessors
                int   src_node         =   0;  // source vertex to start
                int   num_gpus         =   1;  // number of GPUs
                int   delta_factor     =   1;  // default delta_factor = 1
                int   max_grid_size    =   0;  // leave it up to the enactor
                float max_queue_sizing = 1.0;  // default maximum queue sizing

                // determine source vertex to start sssp
                switch (config.src_mode) {
                case randomize: {
                    src_node = graphio::RandomNode(csr_graph.nodes);
                    break;
                }
                case largest_degree: {
                    int max_deg = 0;
                    src_node = csr_graph.GetNodeWithHighestDegree(max_deg);
                    break;
                }
                case manually: {
                    src_node = config.src_node;
                    break;
                }
                default: {
                    src_node = 0;
                    break;
                }
                }
                mark_pred        = config.mark_pred;
                delta_factor     = config.delta_factor;
                max_queue_sizing = config.queue_size;

                switch (mark_pred) {
                case true: {
                    run_sssp<int, unsigned int, int, true>(
                        graph_o,
                        (int*)predecessor,
                        csr_graph,
                        src_node,
                        max_grid_size,
                        max_queue_sizing,
                        num_gpus,
                        delta_factor,
                        context);
                    break;
                }
                case false: {
                    run_sssp<int, unsigned int, int, false>(
                        graph_o,
                        (int*)predecessor,
                        csr_graph,
                        src_node,
                        max_grid_size,
                        max_queue_sizing,
                        num_gpus,
                        delta_factor,
                        context);
                    break;
                }
                }
                // reset for free memory
                csr_graph.row_offsets    = NULL;
                csr_graph.column_indices = NULL;
                csr_graph.edge_values    = NULL;
                break;
            }
            case VALUE_FLOAT: {
                // template type = <int, float, int>
                // not support yet
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/**
 * @brief run_sssp entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[out] graph_o     GRGraph type output
 * @param[out] predecessor Return predeessor if mark_pred = true
 * @param[in]  graph_i     GRGraph type input graph
 * @param[in]  config      Primitive specific configurations
 * @param[in]  data_t      Data type configurations
 */
void gunrock_sssp(
    GRGraph       *graph_o,
    void          *predecessor,
    const GRGraph *graph_i,
    const GRSetup config,
    const GRTypes data_t) {
    unsigned int device = 0;
    device = config.device;
    ContextPtr context = mgpu::CreateCudaDevice(device);
    dispatch_sssp(graph_o, predecessor, graph_i, config, data_t, *context);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
