// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_pr.cu
 *
 * @brief Gunrock Computing Pagerank Implementation
 */

#include <stdio.h>
#include <gunrock/gunrock.h>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/pr/pr_enactor.cuh>
#include <gunrock/app/pr/pr_problem.cuh>
#include <gunrock/app/pr/pr_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::pr;


/******************************************************************************
 * Defines, constants, globals
 ******************************************************************************/
static bool g_verbose;
//static bool g_undirected;
//static bool g_quick;
static bool g_stream_from_host;

template <typename VertexId, typename Value>
struct RankPair {
    VertexId vertex_id;
    Value    page_rank;
    RankPair(VertexId vertex_id, Value page_rank) : vertex_id(vertex_id), page_rank(page_rank) {}
};

template<typename RankPair>
__inline__ bool PRCompare(
    RankPair elem1,
    RankPair elem2)
{
    return elem1.page_rank > elem2.page_rank;
}

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
/*
static void Usage()
{
    printf(
        "\ntest_pr <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--quick] "
        "[--v]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --quick If set will skip the CPU validation code.\n"
        );
}
*/

/**
 * @brief Displays the BFS result (i.e., distance from source)
 *
 * @param[in] source_path Search depth from the source for each node.
 * @param[in] nodes Number of nodes in the graph.
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplaySolution(VertexId *node_id, Value *rank, SizeT nodes)
{
    printf("\nFirst %d labels of the GPU result.", nodes);
    // Print out at most top 10 largest
    int top = (nodes < 10) ? nodes : 10;
    printf("Top %d Page Ranks:\n", top);
    for (int i = 0; i < top; ++i)
    {
        printf("Vertex ID: %d, Page Rank: %5f\n", node_id[i], rank[i]);
    }
}

/**
* Performance/Evaluation statistics
*/
struct Stats {
    const char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;
    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(const char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] h_rank Host-side vector stores computed page rank values for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    Value               *h_rank,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    long long           total_queued,
    double              avg_duty)
{
    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display the specific sample statistics
    printf(" elapsed: %.3f ms", elapsed);
    if (avg_duty != 0)
    {
        printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
    }
    printf("\n");
}

/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/
/**
 * @brief Run PR tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] delta Delta value for computing PageRank, usually set to .85
 * @param[in] error Error threshold value
 * @param[in] max_iter Max iteration for Page Rank computing
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void run_page_rank(
    GunrockGraph *ggraph_out,
    VertexId     *node_ids,
    Value        *page_rank,
    const Csr<VertexId, Value, SizeT> &graph,
    VertexId     source,
    Value        delta,
    Value        error,
    SizeT        max_iter,
    int          max_grid_size,
    int          num_gpus,
    CudaContext& context)
{
    typedef PRProblem<
        VertexId,
        SizeT,
        Value> Problem;

    // Allocate host-side label array for gpu-computed results
    //Value    *h_rank    = (Value*)malloc(sizeof(Value) * graph.nodes);
    //VertexId *h_node_id = (VertexId*)malloc(sizeof(VertexId) * graph.nodes);

    // Allocate BFS enactor map
    PREnactor<false> pr_enactor(g_verbose);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;
    util::GRError(csr_problem->Init(
        g_stream_from_host,
        graph,
        num_gpus),
    "Problem PR Initialization Failed", __FILE__, __LINE__);

    Stats *stats = new Stats("GPU PageRank");

    long long total_queued = 0;
    double    avg_duty = 0.0;

    // Perform BFS
    GpuTimer gpu_timer;

    util::GRError(csr_problem->Reset(
        source, delta, error, pr_enactor.GetFrontierType()),
        "PR Problem Data Reset Failed", __FILE__, __LINE__);
    gpu_timer.Start();
    util::GRError(pr_enactor.template Enact<Problem>(
        context, csr_problem, max_iter, max_grid_size),
        "PR Problem Enact Failed", __FILE__, __LINE__);
    gpu_timer.Stop();

    pr_enactor.GetStatistics(total_queued, avg_duty);
    float elapsed = gpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(csr_problem->Extract(page_rank, node_ids),
        "PageRank Problem Data Extraction Failed", __FILE__, __LINE__);

    // Display Solution
    //DisplaySolution(node_ids, page_rank, graph.nodes);

    DisplayStats(
        *stats,
        page_rank,
        graph,
        elapsed,
        total_queued,
        avg_duty);

    // Cleanup
    delete stats;
    if (csr_problem) delete csr_problem;
    //if (h_rank) free(h_rank);

    hipDeviceSynchronize();
}

/**
 * @brief run_page_rank entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
void gunrock_pr(
    GunrockGraph       *ggraph_out,
    void               *node_ids,
    void               *page_rank,
    const GunrockGraph *ggraph_in,
    GunrockConfig      pr_config,
    GunrockDataType    data_type)
{
    // moderngpu preparations
    int device = 0;
    device = pr_config.device;
    ContextPtr context = mgpu::CreateCudaDevice(device);

    // build input csr format graph
    Csr<int, float, int> csr_graph(false);
    csr_graph.nodes = ggraph_in->num_nodes;
    csr_graph.edges = ggraph_in->num_edges;
    csr_graph.row_offsets    = (int*)ggraph_in->row_offsets;
    csr_graph.column_indices = (int*)ggraph_in->col_indices;

    // page rank configurations
    float delta         = 0.85f; //!< use whatever the specified graph-type's default is
    float error         = 0.01f; //!< error threshold
    int   max_iter      = 20;    //!< maximum number of iterations
    int   max_grid_size = 0;     //!< maximum grid size (0: leave it up to the enactor)
    int   num_gpus      = 1;     //!< number of GPUs for multi-gpu enactor to use
    int   source        = -1;    //!< source node to start

    delta    = pr_config.delta;
    error    = pr_config.error;
    source   = pr_config.source;
    max_iter = pr_config.max_iter;

    run_page_rank<int, float, int>(
        ggraph_out,
        (int*)node_ids,
        (float*)page_rank,
        csr_graph,
        source,
        delta,
        error,
        max_iter,
        max_grid_size,
        num_gpus,
        *context);

    // reset for free memory
    csr_graph.row_offsets    = NULL;
    csr_graph.column_indices = NULL;
    csr_graph.row_offsets    = NULL;
    csr_graph.column_indices = NULL;
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
