// ----------------------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------------------

/**
 * @file pr_app.cu
 *
 * @brief Gunrock PageRank application
 */

#include <gunrock/gunrock.h>

// graph construction utilities
#include <gunrock/graphio/market.cuh>

// page-rank includes
#include <gunrock/app/pr/pr_enactor.cuh>
#include <gunrock/app/pr/pr_problem.cuh>
#include <gunrock/app/pr/pr_functor.cuh>

#include <moderngpu.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::pr;

struct Test_Parameter : gunrock::app::TestParameter_Base {
  public:
    float    delta          ;  // Delta value for PageRank
    float    error          ;  // Error threshold PageRank
    int      max_iter       ;  // Maximum number of iteration

    Test_Parameter() {
        delta    = 0.85f;
        error    = 0.01f;
        max_iter =    50;
        src      =    -1;
    }
    ~Test_Parameter() {
    }
};

template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void runPageRank(GRGraph *output, Test_Parameter *parameter);

template <
    typename      VertexId,
    typename      Value,
    typename      SizeT,
    bool          INSTRUMENT,
    bool          DEBUG >
void sizeCheckPageRank(GRGraph *output, Test_Parameter *parameter) {
    if (parameter->size_check)
        runPageRank<VertexId, Value, SizeT, INSTRUMENT, DEBUG,
                    true > (output, parameter);
    else
        runPageRank<VertexId, Value, SizeT, INSTRUMENT, DEBUG,
                    false> (output, parameter);
}

template <
    typename    VertexId,
    typename    Value,
    typename    SizeT,
    bool        INSTRUMENT >
void debugPageRank(GRGraph *output, Test_Parameter *parameter) {
    if (parameter->debug)
        sizeCheckPageRank<VertexId, Value, SizeT, INSTRUMENT,
                          true > (output, parameter);
    else
        sizeCheckPageRank<VertexId, Value, SizeT, INSTRUMENT,
                          false> (output, parameter);
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 * @param[in] context CudaContext pointer for moderngpu APIs
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT >
void runPageRank(GRGraph *output, Test_Parameter* parameter) {
    if (parameter->instrumented)
        debugPageRank<VertexId, Value, SizeT,  true>(output, parameter);
    else
        debugPageRank<VertexId, Value, SizeT, false>(output, parameter);
}

/**
 * @brief Run PR tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] src Source node for personalized PageRank (if any)
 * @param[in] delta Delta value for computing PageRank, usually set to .85
 * @param[in] error Error threshold value
 * @param[in] max_iter Max iteration for Page Rank computing
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] iterations Number of iterations for running the test
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT,
    bool DEBUG,
    bool SIZE_CHECK >
void runPageRank(GRGraph *output, Test_Parameter *parameter) {
    typedef PRProblem < VertexId,
            SizeT,
            Value > PrProblem;

    typedef PREnactor < PrProblem,
            INSTRUMENT,
            DEBUG,
            SIZE_CHECK > PrEnactor;

    Csr<VertexId, Value, SizeT>
    *graph              = (Csr<VertexId, Value, SizeT>*)parameter->graph;
    int           max_grid_size      = parameter -> max_grid_size;
    int           num_gpus           = parameter -> num_gpus;
    double        max_queue_sizing   = parameter -> max_queue_sizing;
    double        max_in_sizing      = parameter -> max_in_sizing;
    ContextPtr   *context            = (ContextPtr*)parameter -> context;
    std::string   partition_method   = parameter -> partition_method;
    int          *gpu_idx            = parameter -> gpu_idx;
    hipStream_t *streams            = parameter -> streams;
    float         partition_factor   = parameter -> partition_factor;
    int           partition_seed     = parameter -> partition_seed;
    bool          g_stream_from_host = parameter -> g_stream_from_host;
    VertexId      src                = parameter -> src;
    Value         delta              = parameter -> delta;
    Value         error              = parameter -> error;
    SizeT         max_iter           = parameter -> max_iter;
    int           traversal_mode     = parameter -> traversal_mode;
    size_t       *org_size           = new size_t  [num_gpus];
    // Allocate host-side label arrays
    Value        *h_rank             = new Value   [graph->nodes];
    VertexId     *h_node_id          = new VertexId[graph->nodes];

    for (int gpu = 0; gpu < num_gpus; gpu++) {
        size_t dummy;
        hipSetDevice(gpu_idx[gpu]);
        hipMemGetInfo(&(org_size[gpu]), &dummy);
    }

    PrEnactor* enactor = new PrEnactor(num_gpus, gpu_idx);  // enactor map
    PrProblem *problem = new PrProblem;  // Allocate problem on GPU

    util::GRError(
        problem->Init(
            g_stream_from_host,
            graph,
            NULL,
            num_gpus,
            gpu_idx,
            partition_method,
            streams,
            max_queue_sizing,
            max_in_sizing,
            partition_factor,
            partition_seed),
        "Problem pr Initialization Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Init(context, problem, traversal_mode, max_grid_size),
        "PR Enactor Init failed", __FILE__, __LINE__);

    // Perform PageRank
    CpuTimer cpu_timer;

    util::GRError(
        problem->Reset(src, delta, error, max_iter,
                       enactor->GetFrontierType(), max_queue_sizing),
        "pr Problem Data Reset Failed", __FILE__, __LINE__);
    util::GRError(
        enactor->Reset(), "PR Enactor Reset Reset failed", __FILE__, __LINE__);

    printf("_________________________________________\n"); fflush(stdout);
    cpu_timer.Start();
    util::GRError(
        enactor->Enact(traversal_mode),
        "PR Problem Enact Failed", __FILE__, __LINE__);
    cpu_timer.Stop();
    printf("-----------------------------------------\n"); fflush(stdout);
    float elapsed = cpu_timer.ElapsedMillis();

    // Copy out results
    util::GRError(
        problem->Extract(h_rank, h_node_id),
        "PageRank Problem Data Extraction Failed", __FILE__, __LINE__);

    float total_pr = 0;
    for (int i = 0; i < graph->nodes; ++i) {
        total_pr += h_rank[i];
    }
    printf(" Total rank : %lf\n", total_pr);

    output->node_value1 = (Value*)&h_rank[0];
    output->node_value2 = (VertexId*)&h_node_id[0];

    printf(" GPU PageRank finished in %lf msec.\n", elapsed);

    // Clean up
    if (org_size) { delete org_size; org_size = NULL; }
    if (problem ) { delete problem ; problem  = NULL; }
    if (enactor ) { delete enactor ; enactor  = NULL; }
}

/**
 * @brief dispatch function to handle data_types
 *
 * @param[out] graph_o    output of pr problem
 * @param[out] node_ids   output of pr problem
 * @param[out] page_rank  output of pr problem
 * @param[in]  graph_i    GRGraph type input graph
 * @param[in]  config     specific configurations
 * @param[in]  data_t     data type configurations
 * @param[in]  context    moderngpu context
 */
void dispatchPageRank(
    GRGraph       *graph_o,
    const GRGraph *graph_i,
    const GRSetup  config,
    const GRTypes  data_t,
    ContextPtr*    context,
    hipStream_t*  streams) {
    Test_Parameter *parameter = new Test_Parameter;
    parameter->context      =  context;
    parameter->streams      =  streams;
    parameter->num_gpus     = config.num_devices;
    parameter->gpu_idx      = config.device_list;
    parameter->delta        = config.pagerank_delta;
    parameter->error        = config.pagerank_error;
    parameter->max_iter     = config.max_iters;
    parameter->g_undirected = true;

    switch (data_t.VTXID_TYPE) {
    case VTXID_INT: {
        switch (data_t.SIZET_TYPE) {
        case SIZET_INT: {
            switch (data_t.VALUE_TYPE) {
            case VALUE_INT: {  // template type = <int, int, int>
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_UINT: {  // template type = <int, uint, int>
                printf("Not Yet Support This DataType Combination.\n");
                break;
            }
            case VALUE_FLOAT: {  // template type = <int, float, int>
                // build input csr format graph
                Csr<int, int, int> csr(false);
                csr.nodes = graph_i->num_nodes;
                csr.edges = graph_i->num_edges;
                csr.row_offsets    = (int*)graph_i->row_offsets;
                csr.column_indices = (int*)graph_i->col_indices;
                parameter->graph = &csr;

                runPageRank<int, float, int>(graph_o, parameter);

                // reset for free memory
                csr.row_offsets    = NULL;
                csr.column_indices = NULL;
                break;
            }
            }
            break;
        }
        }
        break;
    }
    }
}

/**
 * @brief run_pr entry
 *
 * @param[out] graph_o    output of pr problem
 * @param[out] node_ids   output of pr problem
 * @param[out] page_rank  output of pr problem
 * @param[in]  graph_i    input graph need to process on
 * @param[in]  config     gunrock primitive specific configurations
 * @param[in]  data_t     gunrock data_t struct
 */
void gunrock_pagerank(
    GRGraph       *graph_o,
    const GRGraph *graph_i,
    const GRSetup  config,
    const GRTypes  data_t) {
    // GPU-related configurations
    int           num_gpus =    0;
    int           *gpu_idx = NULL;
    ContextPtr    *context = NULL;
    hipStream_t  *streams = NULL;

    num_gpus = config.num_devices;
    gpu_idx  = new int [num_gpus];
    for (int i = 0; i < num_gpus; ++i) {
        gpu_idx[i] = config.device_list[i];
    }

    // Create streams and MordernGPU context for each GPU
    streams = new hipStream_t[num_gpus * num_gpus * 2];
    context = new ContextPtr[num_gpus * num_gpus];
    printf(" using %d GPUs:", num_gpus);
    for (int gpu = 0; gpu < num_gpus; ++gpu) {
        printf(" %d ", gpu_idx[gpu]);
        util::SetDevice(gpu_idx[gpu]);
        for (int i = 0; i < num_gpus * 2; ++i) {
            int _i = gpu * num_gpus * 2 + i;
            util::GRError(hipStreamCreate(&streams[_i]),
                          "hipStreamCreate fialed.", __FILE__, __LINE__);
            if (i < num_gpus) {
                context[gpu * num_gpus + i] =
                    mgpu::CreateCudaDeviceAttachStream(gpu_idx[gpu],
                                                       streams[_i]);
            }
        }
    }
    printf("\n");

    dispatchPageRank(graph_o, graph_i, config, data_t, context, streams);
}

/*
 * @brief Simple interface take in CSR arrays as input
 * @param[out] pagerank    Return PageRank scores per node
 * @param[in]  num_nodes   Number of nodes of the input graph
 * @param[in]  num_edges   Number of edges of the input graph
 * @param[in]  row_offsets CSR-formatted graph input row offsets
 * @param[in]  col_indices CSR-formatted graph input column indices
 * @param[in]  source      Source to begin traverse
 */
void pagerank(
    int*                node_ids,
    float*              pagerank,
    const int           num_nodes,
    const int           num_edges,
    const int*          row_offsets,
    const int*          col_indices) {
    struct GRTypes data_t;            // primitive-specific data types
    data_t.VTXID_TYPE = VTXID_INT;    // integer
    data_t.SIZET_TYPE = SIZET_INT;    // integer
    data_t.VALUE_TYPE = VALUE_FLOAT;  // float ranks

    struct GRSetup config;            // primitive-specific configures
    int list[] = {0, 1, 2, 3};        // device to run algorithm
    config.num_devices = sizeof(list) / sizeof(list[0]);  // number of devices
    config.device_list    =  list;    // device list to run algorithm
    config.pagerank_delta = 0.85f;    // default delta value
    config.pagerank_error = 0.01f;    // default error threshold
    config.max_iters      =    50;    // maximum number of iterations
    config.top_nodes      =    10;    // number of top nodes

    struct GRGraph *graph_o = (struct GRGraph*)malloc(sizeof(struct GRGraph));
    struct GRGraph *graph_i = (struct GRGraph*)malloc(sizeof(struct GRGraph));

    graph_i->num_nodes   = num_nodes;
    graph_i->num_edges   = num_edges;
    graph_i->row_offsets = (void*)&row_offsets[0];
    graph_i->col_indices = (void*)&col_indices[0];

    printf(" loaded %d nodes and %d edges\n", num_nodes, num_edges);

    gunrock_pagerank(graph_o, graph_i, config, data_t);
    memcpy(pagerank, (float*)graph_o->node_value1, num_nodes * sizeof(float));
    memcpy(node_ids, (  int*)graph_o->node_value2, num_nodes * sizeof(  int));

    if (graph_i) free(graph_i);
    if (graph_o) free(graph_o);
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
