// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_bc.cu
 *
 * @brief Simple test driver program for BFS.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BC includes
#include <gunrock/app/bc/bc_enactor.cuh>
#include <gunrock/app/bc/bc_problem.cuh>
#include <gunrock/app/bc/bc_functor.cuh>

// Operator includes
#include <gunrock/oprtr/edge_map_forward/kernel.cuh>
#include <gunrock/oprtr/vertex_map/kernel.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::bc;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_bc <graph type> <graph type args> [--device=<device_index>] "
        "[--instrumented] [--src=<source index>] [--quick] "
        "[--num_gpus=<gpu number>] [--queue-sizing=<scale factor>]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "--src=<source index>: When source index is -1, compute BC value for each\n"
        "node. Otherwise, debug the delta value for one node\n"
        );
 }

 /**
  * Displays the BC result (sigma value and BC value)
  */
 template<typename Value, typename SizeT>
 void DisplaySolution(Value *sigmas, Value *bc_values, SizeT nodes)
 {
     if (nodes < 20) {
         printf("[");
         for (SizeT i = 0; i < nodes; ++i) {
             PrintValue(i);
             printf(":");
             PrintValue(sigmas[i]);
             printf(",");
             PrintValue(bc_values[i]);
             printf(" ");
         }
         printf("]\n");
     }
 }

 /**
  * Performance/Evaluation statistics
  */

 struct Statistic
 {
    double mean;
    double m2;
    int count;

    Statistic() : mean(0.0), m2(0.0), count(0) {}

    /**
     * Updates running statistic, returning bias-corrected sample variance.
     * Online method as per Knuth.
     */
    double Update(double sample)
    {
        count++;
        double delta = sample - mean;
        mean = mean + (delta / count);
        m2 = m2 + (delta * (sample - mean));
        return m2 / (count - 1);                //bias-corrected
    }
};

/******************************************************************************
 * BC Testing Routines
 *****************************************************************************/

 /**
  * A simple CPU-based reference BC ranking implementation.
  */
 template<
    typename VertexId,
    typename Value,
    typename SizeT>
void RefCPUBC(
    const Csr<VertexId, Value, SizeT>       &graph,
    Value                                   *sigmas,
    Value                                   *bc_values,
    VertexId                                src)
{
    //
    //Perform BC
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    printf("CPU BC finished in %lf msec.", elapsed);
}

/**
 * Run tests
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    VertexId src,
    int max_grid_size,
    int num_gpus,
    double max_queue_sizing)
{
    typedef BCProblem<
        VertexId,
        SizeT,
        Value,
        io::ld::cg,
        io::ld::NONE,
        io::ld::NONE,
        io::ld::cg,
        io::ld::NONE,
        io::st::cg> Problem;

    typedef ForwardFunctor<
        VertexId,
        SizeT,
        Value,
        Problem> FFunctor;
    typedef BackwardFunctor<
        VertexId,
        SizeT,
        Value,
        Problem> BFunctor;


        // Allocate host-side array (for both reference and gpu-computed results)
        Value       *reference_bc_values        = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value       *reference_sigmas           = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value       *h_sigmas                   = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value       *h_bc_values                = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value       *reference_check_sigmas     = (g_quick) ? NULL : reference_sigmas;
        Value       *reference_check_bc_values  = (g_quick) ? NULL : reference_bc_values;

        // Allocate BC enactor map
        BCEnactor<INSTRUMENT> bc_enactor(g_verbose);

        printf("edge: %d\n", graph.edges);

        // Allocate problem on GPU
        Problem *csr_problem = new Problem;
        if (csr_problem->Init(
            g_stream_from_host,
            graph.nodes,
            graph.edges,
            graph.row_offsets,
            graph.column_indices,
            num_gpus)) exit(1);

        //
        // Compute reference CPU BC solution for source-distance
        //
        if (reference_check_bc_values != NULL)
        {
            printf("compute ref value\n");
            RefCPUBC(
                    graph,
                    reference_check_sigmas,
                    reference_check_bc_values,
                    src);
            printf("\n");
        }

        hipError_t         retval = hipSuccess;

        // Perform BFS
        GpuTimer gpu_timer;

        VertexId start_src;
        VertexId end_src;
        if (src == -1)
        {
            start_src = 0;
            end_src = graph.nodes;
        }
        else
        {
            start_src = src;
            end_src = src+1;
        }


        for (VertexId i = start_src; i < end_src; ++i)
        {
            if (retval = csr_problem->Reset(i, bc_enactor.GetFrontierType(), max_queue_sizing)) exit(1);
            gpu_timer.Start();
            if (retval = bc_enactor.template Enact<Problem, FFunctor, BFunctor>(csr_problem, i, max_grid_size)) exit(1);
            gpu_timer.Stop();

            if (retval && (retval != hipErrorInvalidDeviceFunction)) {
                exit(1);
            }
        }

        float elapsed = gpu_timer.ElapsedMillis();

        // Copy out results
        if (csr_problem->Extract(h_sigmas, h_bc_values)) exit(1);

        // Verify the result
        if (reference_check_bc_values != NULL) {
            printf("Validity: ");
            CompareResults(h_bc_values, reference_check_bc_values, graph.nodes, true);
        }
        
        // Display Solution
        DisplaySolution(h_sigmas, h_bc_values, graph.nodes);


        // Cleanup
        if (csr_problem) delete csr_problem;
        if (reference_bc_values) free(reference_bc_values);
        if (reference_sigmas) free(reference_sigmas);
        if (h_sigmas) free(h_sigmas);
        if (h_bc_values) free(h_bc_values);

        hipDeviceSynchronize();
}

template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args)
{
    VertexId            src                 = -1;           // Use whatever the specified graph-type's default is
    std::string         src_str;
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use
    double              max_queue_sizing    = 1.3;          // Maximum size scaling factor for work queues (e.g., 1.0 creates n and m-element vertex and edge frontiers).

    instrumented = args.CheckCmdLineFlag("instrumented");
    args.GetCmdLineArgument("src", src_str);
    if (src_str.empty()) {
        src = 0;
    } else {
        args.GetCmdLineArgument("src", src);
    }

    g_quick = args.CheckCmdLineFlag("quick");
    args.GetCmdLineArgument("num-gpus", num_gpus);
    args.GetCmdLineArgument("queue-sizing", max_queue_sizing);
    g_verbose = args.CheckCmdLineFlag("v");

    if (instrumented) {
            RunTests<VertexId, Value, SizeT, true>(
                graph,
                src,
                max_grid_size,
                num_gpus,
                max_queue_sizing);
    } else {
            RunTests<VertexId, Value, SizeT, false>(
                graph,
                src,
                max_grid_size,
                num_gpus,
                max_queue_sizing);
    }

}



/******************************************************************************
 * Main
 ******************************************************************************/

int main( int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
		Usage();
		return 1;
	}

	DeviceInit(args);
	hipSetDeviceFlags(hipDeviceMapHost);

	//srand(0);									// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = false;

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) {
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") {

		// Matrix-market coordinate-formatted graph file

		typedef int VertexId;							// Use as the node identifier type
		typedef float Value;								// Use as the value type
		typedef int SizeT;								// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);         // default value for stream_from_host is false

		if (graph_args < 1) { Usage(); return 1; }
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		if (graphio::BuildMarketGraph<false>(
			market_filename, 
			csr, 
			g_undirected) != 0) 
		{
			return 1;
		}

        csr.DisplayGraph();
        fflush(stdout);

		// Run tests
		RunTests(csr, args);

	} else {

		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;

	}

	return 0;
}
