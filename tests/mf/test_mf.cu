#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_mf.cu
 *
 * @brief Simple test driver program for max-flow algorithm.
 */

#include <gunrock/app/mf/mf_app.cu>
#include <gunrock/app/test_base.cuh>

#define debug_aml(a...) std::cerr << __FILE__ << ":" << __LINE__ << " " << \
    a << "\n";

using namespace gunrock;

/*****************************************************************************
* Main
*****************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT	  Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use int as the value type
    hipError_t operator()(util::Parameters &parameters, VertexT v, SizeT s, 
	    ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT, 
	  graph::HAS_EDGE_VALUES | graph::HAS_CSR> GraphT;
	typedef typename GraphT::CsrT CsrT;
        hipError_t retval = hipSuccess;
	bool quick = parameters.Get<bool>("quick");
        bool quiet = parameters.Get<bool>("quiet");
	
	//
	// Load Graph
	//
        GraphT graph;
        util::CpuTimer cpu_timer; cpu_timer.Start();
	debug_aml("Start Load Graph");
        GUARD_CU(graphio::LoadGraph(parameters, graph));
        
	//FOR DEBUG: force edge values to be 1
        /*for (SizeT e=0; e < graph.edges; e++){
	    graph.CsrT::edge_values[e] = 2;
	}*/

	if (parameters.Get<VertexT>("source") == 
		util::PreDefinedValues<VertexT>::InvalidValue){
	    parameters.Set("source", 0);
	}
	if (parameters.Get<VertexT>("sink") == 
		util::PreDefinedValues<VertexT>::InvalidValue){
	    parameters.Set("sink", graph.nodes-1);
	}
        
	cpu_timer.Stop();
        
	parameters.Set("load-time", cpu_timer.ElapsedMillis());
	debug_aml("load-time is " << cpu_timer.ElapsedMillis());

	VertexT source = parameters.Get<VertexT>("source");
	VertexT sink = parameters.Get<VertexT>("sink");

	//
        // Compute reference CPU max flow algorithm.
	//
        ValueT max_flow;
	ValueT* flow_edges = (ValueT*)malloc(sizeof(ValueT)*graph.edges);
	VertexT* reverse = (VertexT*)malloc(sizeof(VertexT)*graph.edges);
	
        util::PrintMsg("______CPU reference algorithm______", true);
	double elapsed = app::mf::CPU_Reference
	    (parameters, graph, source, sink, max_flow, reverse, flow_edges);
        util::PrintMsg("------------------------------------\n\elapsed: " + 
		std::to_string(elapsed) + " ms, max flow = " +
		std::to_string(max_flow), true);

	
        std::vector<std::string> switches{"advance-mode"};
	GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
	[flow_edges, reverse](util::Parameters &parameters, GraphT &graph)
	{
	debug_aml("go to RunTests");
	return app::mf::RunTests(parameters, graph, reverse, flow_edges);
	}));

	// Clean up
	free(flow_edges);
	
        return retval;
    }
};

int main(int argc, char** argv)
{
    debug_aml("Main: start");
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test mf");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::mf::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());
    debug_aml("Main: parameters checked - ok");

    return app::Switch_Types<
        app::VERTEXT_U32B | 
        app::SIZET_U32B | 
        app::VALUET_U32B | 
	app::UNDIRECTED >
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:

