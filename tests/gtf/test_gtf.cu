#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_gtf.cu
 *
 * @brief Simple test driver program for max-flow algorithm.
 */

#include <gunrock/app/gtf/gtf_app.cu>
#include <gunrock/app/test_base.cuh>

#define debug_aml(a...)
//#define debug_aml(a...) {printf(a); printf("\n");}

using namespace gunrock;

/*****************************************************************************
* Main
*****************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct
{
    /**
     * @brief the actual main function, after type switching
     * @tparam VertexT	  Type of vertex identifier
     * @tparam SizeT      Type of graph size, i.e. type of edge identifier
     * @tparam ValueT     Type of edge values
     * @param  parameters Command line parameters
     * @param  v,s,val    Place holders for type deduction
     * \return hipError_t error message(s), if any
     */
    template <
        typename VertexT, // Use int as the vertex identifier
        typename SizeT,   // Use int as the graph size type
        typename ValueT>  // Use int as the value type
    hipError_t operator()(util::Parameters &parameters, VertexT v, SizeT s,
	    ValueT val)
    {
        typedef typename app::TestGraph<VertexT, SizeT, ValueT,
	       graph::HAS_EDGE_VALUES | graph::HAS_CSR> GraphT;
	    typedef typename GraphT::CsrT CsrT;
        hipError_t retval = hipSuccess;
	    bool quick = parameters.Get<bool>("quick");
        bool quiet = parameters.Get<bool>("quiet");

    	//
    	// Load Graph
    	//
        util::CpuTimer cpu_timer; cpu_timer.Start();
    	debug_aml("Start Load Graph");
        GraphT u_graph;
    	bool undirected;
    	parameters.Get("undirected", undirected);

    	if (undirected) {
    	    debug_aml("graph is undirected");
            debug_aml("Load undirected graph");
        	//parameters.Set<int>("undirected", 1);
        	parameters.Set<bool>("remove-duplicate-edges", true);
            GUARD_CU(graphio::LoadGraph(parameters, u_graph));

    	} else {
    	    debug_aml("graph is directed");
            GraphT d_graph;
            debug_aml("Load directed graph");
            //parameters.Set<int>("undirected", 0);
    	    //parameters.Set<bool>("remove-duplicate-edges", false);
    	    GUARD_CU(graphio::LoadGraph(parameters, d_graph));

            debug_aml("Directed graph:");
    	    debug_aml("number of edges %d", d_graph.edges);
    	    debug_aml("number of nodes %d", d_graph.nodes);

            GUARD_CU(graphio::MakeUndirected(d_graph, u_graph, false));
            GUARD_CU(mf::CorrectReverseCapacities(
                d_graph.csr(), u_graph.csr()));

            GUARD_CU(d_graph.Release());
        }

        util::Array1D<SizeT, ValueT> weights;
        std::string weights_filename = parameters.Get<std::string>("weights");
        GUARD_CU(weights.Read(weights_filename));

        GraphT graph;
        GUARD_CU(gtf::AddSourceSink(u_graph.csr(), weights, graph.csr()));
        GUARD_CU(u_graph.Release());

    	cpu_timer.Stop();
    	parameters.Set("load-time", cpu_timer.ElapsedMillis());
    	debug_aml("load-time is %lf",cpu_timer.ElapsedMillis());

        GUARD_CU(parameters.Set("source", graph.nodes - 2));
        GUARD_CU(parameters.Set("sink"  , graph.nodes - 1));

    	debug_aml("Undirected graph:");
    	debug_aml("number of edges %d", graph.edges);
    	debug_aml("number of nodes %d", graph.nodes);

        util::Array1D<SizeT, SizeT> reverse_edges;
        reverse_edges.SetName("reverse_edges");
        GUARD_CU(reverse_edges.Allocate(graph.edges, util::HOST));

    	GUARD_CU(mf::InitReverse(graph, reverse_edges));

	    //
        // Compute reference CPU GTF algorithm.
	    //
    	util::PrintMsg("______CPU reference algorithm______", true);
    	double elapsed = app::gtf::CPU_Reference
    	    (parameters, graph, reverse_edges);
        util::PrintMsg("-----------------------------------\n"
            "Elapsed: " + std::to_string(elapsed) + " ms", true);

        std::vector<std::string> switches{"advance-mode"};
    	GUARD_CU(app::Switch_Parameters(parameters, graph, switches,
    	[reverse_edges](util::Parameters &parameters, GraphT &graph)
    	{
    	    //return app::gtf::RunTests(parameters, graph, reverse_edges);
    	}));

    	// Clean up
    	GUARD_CU(reverse_edges.Release());
        GUARD_CU(graph.Release());
        
        return retval;
    }
};

int main(int argc, char** argv)
{
    debug_aml("Main: start");
    hipError_t retval = hipSuccess;
    util::Parameters parameters("test gtf");
    GUARD_CU(graphio::UseParameters(parameters));
    GUARD_CU(app::gtf::UseParameters(parameters));
    GUARD_CU(app::UseParameters_test(parameters));
    GUARD_CU(parameters.Parse_CommandLine(argc, argv));
    if (parameters.Get<bool>("help"))
    {
        parameters.Print_Help();
        return hipSuccess;
    }
    GUARD_CU(parameters.Check_Required());
    debug_aml("Main: parameters checked - ok");

    return app::Switch_Types<
        app::VERTEXT_U32B |
        app::SIZET_U32B |
        app::VALUET_F64B |
	    app::DIRECTED | app::UNDIRECTED >
        (parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
