// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_tmp.cu
 *
 * @brief Simple test driver program for general graph algorithm.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.h>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

using namespace gunrock;
using namespace gunrock::util;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_undirected;
bool g_with_edge_values;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_tmp <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--withvalue]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        );
 }

template <
	typename VertexId,
	typename Value,
	typename SizeT>
void RunTests(
	Csr<VertexId, Value, SizeT> &csr,
	CommandLineArgs &args)
{
	csr.PrintHistogram();
}


/******************************************************************************
 * Main
 ******************************************************************************/

int main( int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
		Usage();
		return 1;
	}

	DeviceInit(args);
	hipSetDeviceFlags(hipDeviceMapHost);

	//srand(0);									// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = args.CheckCmdLineFlag("undirected");
    g_with_edge_values = args.CheckCmdLineFlag("withvalue");

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) {
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") {

		// Matrix-market coordinate-formatted graph file

		typedef int VertexId;							// Use as the node identifier type
		typedef int Value;								// Use as the value type
		typedef int SizeT;								// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);         // default value for stream_from_host is false

		if (graph_args < 1) { Usage(); return 1; }
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		if (graphio::BuildMarketGraph<false>(
			market_filename, 
			csr, 
			g_undirected) != 0) 
		{
			return 1;
		}

		// Run tests
		RunTests(csr, args);

	} else {

		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;

	}

	return 0;
}
