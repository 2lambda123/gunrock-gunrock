// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_mst.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <utility>
#include <iostream>
#include <cstdlib>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/mst/mst_enactor.cuh>
#include <gunrock/app/mst/mst_problem.cuh>
#include <gunrock/app/mst/mst_functor.cuh>

// Operator includes
#include <gunrock/oprtr/edge_map_forward/kernel.cuh>
#include <gunrock/oprtr/vertex_map/kernel.cuh>

#include <moderngpu.cuh>

#include <boost/config.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/prim_minimum_spanning_tree.hpp>


using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::mst;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_mst <graph type> <graph type args> [--device=<device_index>] "
        "[--instrumented] [--quick] "
        "[--v]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --quick If set will skip the CPU validation code.\n"
        );
 }

 /**
  * @brief Displays the MST result
  *
  */
 template<typename Value, typename SizeT>
 void DisplaySolution()
 { 
 }

 
/******************************************************************************
 * MST Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference MST implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  *
  * @param[in] graph Reference to the CSR graph we process on
  */
 template<
    typename VertexId,
    typename Value,
    typename SizeT>
void SimpleReferenceMST(
    Value                                   *weights,
    const Csr<VertexId, Value, SizeT>       &graph)
{
    //Preparation
    using namespace boost;
    typedef adjacency_list < vecS, vecS, undirectedS,
            property<vertex_distance_t, int>, property < edge_weight_t, int > > Graph;
    typedef std::pair < int, int > E;
    int num_nodes = graph.nodes;
    int num_edges = graph.edges;
    E *edge_pairs = new E[num_edges];
    int idx = 0;
    printf("node %d edge %d\n", num_nodes, num_edges);

    for (int i = 0; i < num_nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
        {
            edge_pairs[idx++] = std::make_pair(i, graph.column_indices[j]);
        }
    }
    /*Graph g(num_nodes);
  property_map<Graph, edge_weight_t>::type weightmap = get(edge_weight, g); 
  for (std::size_t j = 0; j < sizeof(edge_pairs) / sizeof(E); ++j) {
  printf("%d, %d\n", edge_pairs[j].first, edge_pairs[j].second);
    graph_traits<Graph>::edge_descriptor e; bool inserted;
    tie(e, inserted) = add_edge(edge_pairs[j].first, edge_pairs[j].second, g);
    weightmap[e] = weights[j];
  }*/
    Graph g(edge_pairs, edge_pairs + num_edges, weights, num_nodes);
    property_map<Graph, edge_weight_t>::type weightmap = get(edge_weight, g);
    std::vector < graph_traits < Graph >::vertex_descriptor >
        p(num_vertices(g));

        typedef graph_traits<Graph>::edge_iterator edge_iterator;

std::pair<edge_iterator, edge_iterator> ei = edges(g);
for(edge_iterator edge_iter = ei.first; edge_iter != ei.second; ++edge_iter) {
        std::cout << "(" << source(*edge_iter, g) << ", " << target(*edge_iter, g) << ")\n";
            }

    //
    //compute MST
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();
    prim_minimum_spanning_tree(g, &p[0]);

    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    printf("CPU MST finished in %lf msec.\n", elapsed);

    for (std::size_t i = 0; i != p.size(); ++i)
        if (p[i] != i)
            std::cout << "parent[" << i << "] = " << p[i] << std::endl;
        else
            std::cout << "parent[" << i << "] = no parent" << std::endl;

}

/**
 * @brief Run MST tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    int max_grid_size,
    int num_gpus,
    mgpu::CudaContext& context)
{

    typedef MSTProblem<
        VertexId,
        SizeT,
        Value> Problem;

    // Allocate BFS enactor map
    MSTEnactor<INSTRUMENT> mst_enactor(g_verbose);

    // Allocate problem on GPU
    Problem *csr_problem = new Problem;
    util::GRError(csr_problem->Init(
                g_stream_from_host,
                graph,
                num_gpus), "Problem MST Initialization Failed", __FILE__, __LINE__);

    // Perform MST
    GpuTimer gpu_timer;

        util::GRError(csr_problem->Reset(mst_enactor.GetFrontierType()), "MST Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        util::GRError(mst_enactor.template Enact<Problem>(context, csr_problem, max_grid_size), "MST Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();

        float elapsed = gpu_timer.ElapsedMillis();

        // Copy out results
        // TODO: write the extract function
        //util::GRError(csr_problem->Extract(h_result), "MST Problem Data Extraction Failed", __FILE__, __LINE__);

        // Verify the result
        //SimpleReferenceMST(graph.edge_values, graph);
        
        // Cleanup
        if (csr_problem) delete csr_problem;

        hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args,
    mgpu::CudaContext& context)
{
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use

    instrumented = args.CheckCmdLineFlag("instrumented");

    g_quick = args.CheckCmdLineFlag("quick");
    g_verbose = args.CheckCmdLineFlag("v");

    if (instrumented) {
        RunTests<VertexId, Value, SizeT, true>(
                        graph,
                        max_grid_size,
                        num_gpus,
                        context);
    } else {
        RunTests<VertexId, Value, SizeT, false>(
                        graph,
                        max_grid_size,
                        num_gpus,
                        context);
    }
}



/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
		Usage();
		return 1;
	}

	//DeviceInit(args);
	//hipSetDeviceFlags(hipDeviceMapHost);
	int dev = 0;
    args.GetCmdLineArgument("device", dev);
    mgpu::ContextPtr context = mgpu::CreateCudaDevice(dev);

	//srand(0);									// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = true;

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) {
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") {

		// Matrix-market coordinate-formatted graph file

		typedef int VertexId;							// Use as the node identifier type
		typedef unsigned int Value;								// Use as the value type
		typedef int SizeT;								// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);         // default value for stream_from_host is false

		if (graph_args < 1) { Usage(); return 1; }
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		if (graphio::BuildMarketGraph<true>(
			market_filename, 
			csr, 
			g_undirected,
			false) != 0) // no inverse graph
		{
			return 1;
		}

		csr.DisplayGraph();

        for (int i = 0; i < csr.edges; ++i)
        {
            printf("%d ", csr.edge_values[i]);
        }
        printf("\n");

        Csr<VertexId, Value, SizeT> csr2(false);
        graphio::BuildMarketGraph<true>(
        market_filename,
        csr2,
        false,
        false);

        csr2.DisplayGraph();

        SimpleReferenceMST(csr2.edge_values, csr2);


		// Run tests
		RunTests(csr, args, *context);

	} else {

		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;

	}

	return 0;
}
