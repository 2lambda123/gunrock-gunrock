#include "hip/hip_runtime.h"
// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_pr.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <iostream>
#include <cstdlib>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// BFS includes
#include <gunrock/app/pr/pr_enactor.cuh>
#include <gunrock/app/pr/pr_problem.cuh>
#include <gunrock/app/pr/pr_functor.cuh>

// Operator includes
#include <gunrock/oprtr/advance/kernel.cuh>
#include <gunrock/oprtr/filter/kernel.cuh>

#include <moderngpu.cuh>

// boost includes
#include <boost/config.hpp>
#include <boost/utility.hpp>
#include <boost/graph/adjacency_list.hpp>
#include <boost/graph/page_rank.hpp>


using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::pr;


/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/

bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

template <typename VertexId, typename Value>
struct RankPair {
    VertexId        vertex_id;
    Value           page_rank;

    RankPair(VertexId vertex_id, Value page_rank) : vertex_id(vertex_id), page_rank(page_rank) {}
};

template<typename RankPair>
bool PRCompare(
    RankPair elem1,
    RankPair elem2)
{
    return elem1.page_rank > elem2.page_rank;
}

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
 void Usage()
 {
 printf("\ntest_pr <graph type> <graph type args> [--device=<device_index>] "
        "[--undirected] [--instrumented] [--quick] "
        "[--v]\n"
        "\n"
        "Graph types and args:\n"
        "  market [<file>]\n"
        "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
        "    edges from stdin (or from the optionally-specified file).\n"
        "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
        "  --undirected If set then treat the graph as undirected.\n"
        "  --instrumented If set then kernels keep track of queue-search_depth\n"
        "  and barrier duty (a relative indicator of load imbalance.)\n"
        "  --quick If set will skip the CPU validation code.\n"
        );
 }

 /**
  * @brief Displays the BFS result (i.e., distance from source)
  *
  * @param[in] source_path Search depth from the source for each node.
  * @param[in] nodes Number of nodes in the graph.
  */
 template<typename Value, typename SizeT>
 void DisplaySolution(Value *rank, SizeT nodes)
 { 
     //sort the top page ranks
     RankPair<SizeT, Value> *pr_list = (RankPair<SizeT, Value>*)malloc(sizeof(RankPair<SizeT, Value>) * nodes);
     Value total_pr = 0;
     for (int i = 0; i < nodes; ++i)
     {
         pr_list[i].vertex_id = i;
         pr_list[i].page_rank = rank[i];
         total_pr += rank[i];
     }
     std::stable_sort(pr_list, pr_list + nodes, PRCompare<RankPair<SizeT, Value> >);

     // Print out at most top 10 largest components
     int top = (nodes < 10) ? nodes : 10;
     printf("Top %d Page Ranks:\n", top);
     for (int i = 0; i < top; ++i)
     {
         printf("Vertex ID: %d, Page Rank: %5f\n", pr_list[i].vertex_id, pr_list[i].page_rank);
     }
     printf("total pr: %5f\n", total_pr);

     free(pr_list);
 }

 /**
  * Performance/Evaluation statistics
  */ 

struct Stats {
    char *name;
    Statistic rate;
    Statistic search_depth;
    Statistic redundant_work;
    Statistic duty;

    Stats() : name(NULL), rate(), search_depth(), redundant_work(), duty() {}
    Stats(char *name) : name(name), rate(), search_depth(), redundant_work(), duty() {}
};

/**
 * @brief Displays timing and correctness statistics
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * 
 * @param[in] stats Reference to the Stats object defined in RunTests
 * @param[in] h_rank Host-side vector stores computed page rank values for validation
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] elapsed Total elapsed kernel running time
 * @param[in] total_queued Total element queued in BFS kernel running process
 * @param[in] avg_duty Average duty of the BFS kernels
 */
template<
    typename VertexId,
    typename Value,
    typename SizeT>
void DisplayStats(
    Stats               &stats,
    Value               *h_rank,
    const Csr<VertexId, Value, SizeT> &graph,
    double              elapsed,
    long long           total_queued,
    double              avg_duty)
{
    
    // Display test name
    printf("[%s] finished. ", stats.name);

    // Display the specific sample statistics
    printf(" elapsed: %.3f ms", elapsed);
    if (avg_duty != 0) {
        printf("\n avg CTA duty: %.2f%%", avg_duty * 100);
    }
    printf("\n");
}




/******************************************************************************
 * BFS Testing Routines
 *****************************************************************************/

 /**
  * @brief A simple CPU-based reference Page Rank implementation.
  *
  * @tparam VertexId
  * @tparam Value
  * @tparam SizeT
  *
  * @param[in] graph Reference to the CSR graph we process on
  * @param[in] rank Host-side vector to store CPU computed labels for each node
  * @param[in] delta delta for computing PR
  * @param[in] error error threshold
  * @param[in] max_iter max iteration to go
  */
 template<
    typename VertexId,
    typename Value,
    typename SizeT>
void SimpleReferencePr(
    const Csr<VertexId, Value, SizeT>       &graph,
    Value                                   *rank,
    Value                                   delta,
    Value                                   error,
    SizeT                                   max_iter) 
{
    using namespace boost;

    //Preparation
    typedef adjacency_list<vecS, vecS, bidirectionalS, no_property, property<edge_index_t, int> > Graph;

    Graph g;

    for (int i = 0; i < graph.nodes; ++i)
    {
        for (int j = graph.row_offsets[i]; j < graph.row_offsets[i+1]; ++j)
        {
            Graph::edge_descriptor e =
            add_edge(i, graph.column_indices[j], g).first;
            put(edge_index, g, e, i);
        }
    }

    
    //
    //compute page rank
    //

    CpuTimer cpu_timer;
    cpu_timer.Start();

    remove_dangling_links(g);

    std::vector<Value> ranks(num_vertices(g));
    page_rank(g,
              make_iterator_property_map(ranks.begin(),
              get(boost::vertex_index, g)),
              boost::graph::n_iterations(max_iter));
    
    cpu_timer.Stop();
    float elapsed = cpu_timer.ElapsedMillis();

    for (std::size_t i = 0; i < num_vertices(g); ++i) {
        rank[i] = ranks[i];
    }

    printf("CPU BFS finished in %lf msec.\n", elapsed);
}

/**
 * @brief Run PR tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] delta Delta value for computing PageRank, usually set to .85
 * @param[in] error Error threshold value
 * @param[in] max_iter Max iteration for Page Rank computing
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 * @param[in] context CudaContext for moderngpu to use
 *
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT,
    bool INSTRUMENT>
void RunTests(
    const Csr<VertexId, Value, SizeT> &graph,
    VertexId src,
    Value delta,
    Value error,
    SizeT max_iter,
    int max_grid_size,
    int num_gpus,
    CudaContext& context)
{
    
    typedef PRProblem<
        VertexId,
        SizeT,
        Value> Problem;

        // Allocate host-side label array (for both reference and gpu-computed results)
        Value    *reference_rank       = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value    *h_rank               = (Value*)malloc(sizeof(Value) * graph.nodes);
        Value    *reference_check        = (g_quick) ? NULL : reference_rank;

        // Allocate BFS enactor map
        PREnactor<INSTRUMENT> pr_enactor(g_verbose);

        // Allocate problem on GPU
        Problem *csr_problem = new Problem;
        util::GRError(csr_problem->Init(
            g_stream_from_host,
            graph,
            num_gpus), "Problem pr Initialization Failed", __FILE__, __LINE__); 

        Stats *stats = new Stats("GPU PageRank");

        long long           total_queued = 0;
        double              avg_duty = 0.0;

        // Perform BFS
        GpuTimer gpu_timer;

        util::GRError(csr_problem->Reset(src, delta, error, pr_enactor.GetFrontierType()), "pr Problem Data Reset Failed", __FILE__, __LINE__);
        gpu_timer.Start();
        util::GRError(pr_enactor.template Enact<Problem>(context, csr_problem, max_iter, max_grid_size), "pr Problem Enact Failed", __FILE__, __LINE__);
        gpu_timer.Stop();

        pr_enactor.GetStatistics(total_queued, avg_duty);

        float elapsed = gpu_timer.ElapsedMillis();

        // Copy out results
        util::GRError(csr_problem->Extract(h_rank), "PageRank Problem Data Extraction Failed", __FILE__, __LINE__);

        float total_pr = 0;
        for (int i = 0; i < graph.nodes; ++i)
        {
            total_pr += h_rank[i];
        }

        //
        // Compute reference CPU PR solution for source-distance
        //
        if (reference_check != NULL && total_pr > 0)
        {
            printf("compute ref value\n");
            SimpleReferencePr(
                    graph,
                    reference_check,
                    delta,
                    error,
                    max_iter);
            printf("\n");
        }

        // Verify the result
        if (reference_check != NULL && total_pr > 0) {
            printf("Validity: ");
            CompareResults(h_rank, reference_check, graph.nodes, true);
        }
        printf("\nFirst 40 labels of the GPU result."); 
        // Display Solution
        DisplaySolution(h_rank, graph.nodes);

        DisplayStats(
            *stats,
            h_rank,
            graph,
            elapsed,
            total_queued,
            avg_duty);


        // Cleanup
        delete stats;
        if (csr_problem) delete csr_problem;
        if (reference_check) free(reference_check);
        if (h_rank) free(h_rank);

        hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <
    typename VertexId,
    typename Value,
    typename SizeT>
void RunTests(
    Csr<VertexId, Value, SizeT> &graph,
    CommandLineArgs &args,
    CudaContext& context)
{
    Value               delta               = 0.85f;           // Use whatever the specified graph-type's default is
    Value               error               = 0.01f;        // Error threshold
    SizeT               max_iter            = 20;
    bool                instrumented        = false;        // Whether or not to collect instrumentation from kernels
    int                 max_grid_size       = 0;            // maximum grid size (0: leave it up to the enactor)
    int                 num_gpus            = 1;            // Number of GPUs for multi-gpu enactor to use
    VertexId            src                 = -1;

    instrumented = args.CheckCmdLineFlag("instrumented");
    args.GetCmdLineArgument("delta", delta);
    args.GetCmdLineArgument("error", error);
    args.GetCmdLineArgument("max-iter", max_iter);
    args.GetCmdLineArgument("src", src);

    g_quick = args.CheckCmdLineFlag("quick");
    g_verbose = args.CheckCmdLineFlag("v");

    if (instrumented) {
        RunTests<VertexId, Value, SizeT, true>(
                        graph,
                        src,
                        delta,
                        error,
                        max_iter,
                        max_grid_size,
                        num_gpus,
                        context);
    } else {
        RunTests<VertexId, Value, SizeT, false>(
                        graph,
                        src,
                        delta,
                        error,
                        max_iter,
                        max_grid_size,
                        num_gpus,
                        context);
    }
}



/******************************************************************************
* Main
******************************************************************************/

int main( int argc, char** argv)
{
	CommandLineArgs args(argc, argv);

	if ((argc < 2) || (args.CheckCmdLineFlag("help"))) {
		Usage();
		return 1;
	}

	//DeviceInit(args);
	//hipSetDeviceFlags(hipDeviceMapHost);
	int dev = 0;
    args.GetCmdLineArgument("device", dev);
    ContextPtr context = mgpu::CreateCudaDevice(dev);

	//srand(0);									// Presently deterministic
	//srand(time(NULL));

	// Parse graph-contruction params
	g_undirected = args.CheckCmdLineFlag("undirected");

	std::string graph_type = argv[1];
	int flags = args.ParsedArgc();
	int graph_args = argc - flags - 1;

	if (graph_args < 1) {
		Usage();
		return 1;
	}
	
	//
	// Construct graph and perform search(es)
	//

	if (graph_type == "market") {

		// Matrix-market coordinate-formatted graph file

		typedef int VertexId;							// Use as the node identifier type
		typedef float Value;								// Use as the value type
		typedef int SizeT;								// Use as the graph size type
		Csr<VertexId, Value, SizeT> csr(false);         // default value for stream_from_host is false

		if (graph_args < 1) { Usage(); return 1; }
		char *market_filename = (graph_args == 2) ? argv[2] : NULL;
		if (graphio::BuildMarketGraph<false>(
			market_filename, 
			csr, 
			g_undirected,
			false) != 0) // no inverse graph
		{
			return 1;
		}

		csr.PrintHistogram();

		// Run tests
		RunTests(csr, args, *context);


	} else {

		// Unknown graph type
		fprintf(stderr, "Unspecified graph type\n");
		return 1;

	}

	return 0;
}
