// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * topk_app.cu
 *
 * @brief Simple test driver program for computing Top K
 */

#include <cstdlib>
#include <stdio.h> 
#include <gunrock/app/topk/topk_enactor.cuh>
#include <gunrock/app/topk/topk_problem.cuh>

using namespace gunrock::app::topk;

/**
 * @brief Run TopK
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 *
 */
template <typename VertexId, typename SizeT, typename Value>
void topk_run(const SizeT    row_offsets,
	      const VertexId col_indices,
	      const SizeT    row_offsets,
	      const VertexId col_indices,
	      const SizeT    num_nodes,
	      const SizeT    num_edges,
	      const SizeT    top_nodes,
	      const int      data_type)
{
  
  // define the problem data structure for graph primitive
  typedef TOPKProblem<VertexId, SizeT, Value> Problem;
  
  // INSTRUMENT specifies whether we want to keep such statistical data
  // Allocate TopK enactor map 
  TOPKEnactor<INSTRUMENT> topk_enactor(g_verbose);
  
  // allocate problem on GPU, create a pointer of the TOPKProblem type 
  Problem *topk_problem = new Problem;
  
  // reset top_nodes if input k > total number of nodes
  if (top_nodes > num_nodes) { top_nodes = num_nodes; }
  
  // malloc host memory
  VertexId *h_node_id = (VertexId*)malloc(sizeof(VertexId) * top_nodes);
  Value    *h_degrees = (  Value* )malloc(sizeof(  Value ) * top_nodes);
  
  // copy data from CPU to GPU, initialize data members in DataSlice for graph
  util::GRError(topk_problem->Init(g_stream_from_host,
				   row_offsets,
				   col_indices,
				   row_offsets,
				   col_indices,
				   num_gpus), 
		"Problem TOPK Initialization Failed", __FILE__, __LINE__);
    
  // reset values in DataSlice for graph
  util::GRError(topk_problem->Reset(topk_enactor.GetFrontierType()), 
		"TOPK Problem Data Reset Failed", __FILE__, __LINE__);
  
  // launch topk enactor
  util::GRError(topk_enactor.template Enact<Problem>(context, 
						     topk_problem, 
						     top_nodes, 
						     max_grid_size), 
		"TOPK Problem Enact Failed", __FILE__, __LINE__);
  
  // copy out results back to CPU from GPU using Extract
  util::GRError(topk_problem->Extract(h_node_id,
				      h_degrees,
				      top_nodes),
		"TOPK Problem Data Extraction Failed", __FILE__, __LINE__);
  
  // display solution
  DisplaySolution(h_node_id, h_degrees, top_nodes);
  
  // cleanup if neccessary
  if (topk_problem) { delete topk_problem; }
  if (h_node_id)    {   free(h_node_id);   }
  if (h_degrees)    {   free(h_degrees);   }
  
  hipDeviceSynchronize();
}

void topk_dispatch(const void *row_offsets,
		   const void *col_indices,
		   const void *row_offsets,
		   const void *col_indices,
		   size_t     num_nodes,
		   size_t     num_edges,
		   size_t     top_nodes,
		   const int  data_type);
{
  switch (data_type)
  {
  case (value=unsigned int):
    topk_run<unsigned int, 
	     unsigned int, 
	     unsigned int>((const unsigned int*)row_offsets,
			   (const unsigned int*)col_indices,
			   (const unsigned int*)row_offsets,
			   (const unsigned int*)col_indices,
			   num_nodes, 
			   num_edges, 
			   top_nodes,
			   data_type);
  case (value=double):
    topk_run<unsigned int, 
	     unsigned int, 
	     double>((const unsigned int*)row_offsets,
		     (const unsigned int*)col_indices,
		     (const unsigned int*)row_offsets,
		     (const unsigned int*)col_indices,
		     num_nodes, 
		     num_edges, 
		     top_nodes,
		     data_type);
  }
}

/* end */
