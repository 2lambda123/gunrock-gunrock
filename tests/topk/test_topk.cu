// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_topk.cu
 *
 * @brief Simple test driver program for computing Pagerank.
 */

#include <stdio.h> 
#include <string>
#include <deque>
#include <vector>
#include <utility>
#include <iostream>
#include <cstdlib>
#include <algorithm>
#include <fstream>
#include <map>

// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph construction utils
#include <gunrock/graphio/market.cuh>

// Degree Centrality includes
#include <gunrock/app/topk/topk_enactor.cuh>
#include <gunrock/app/topk/topk_problem.cuh>

// Operator includes
#include <gunrock/oprtr/filter/kernel.cuh>

using namespace gunrock;
using namespace gunrock::util;
using namespace gunrock::oprtr;
using namespace gunrock::app::topk;

/******************************************************************************
 * Defines, constants, globals 
 ******************************************************************************/
bool g_verbose;
bool g_undirected;
bool g_quick;
bool g_stream_from_host;

/******************************************************************************
 * Housekeeping Routines
 ******************************************************************************/
void Usage()
{
  printf("\ntest_topk <graph type> <graph type args> [--top=<K_value>] [--device=<device_index>] "
	 "[--instrumented] [--quick] "
	 "[--v]\n"
	 "\n"
	 "Graph types and args:\n"
	 "  market [<file>]\n"
	 "    Reads a Matrix-Market coordinate-formatted graph of directed/undirected\n"
	 "    edges from stdin (or from the optionally-specified file).\n"
	 "    k value top K value.\n"
	 "  --device=<device_index>  Set GPU device for running the graph primitive.\n"
	 "  --instrumented If set then kernels keep track of queue-search_depth\n"
	 "  and barrier duty (a relative indicator of load imbalance.)\n"
	 "  --quick If set will skip the CPU validation code.\n"
	 );
}

/**
 * @brief displays the top K results
 *
 */
template<typename VertexId, 
	 typename Value, 
	 typename SizeT>
void DisplaySolution(VertexId *h_node_id, 
		     Value    *h_degrees, 
		     SizeT    num_nodes)
{
 
  // at most display first 100 results
  if (num_nodes > 100) 
  { 
    num_nodes = 100; 
  }
  printf("==> top %d centrality nodes:\n", num_nodes);
  for (SizeT i = 0; i < num_nodes; ++i)
  { 
    printf("%d %d\n", h_node_id[i], h_degrees[i]); 
  }
  printf("\n");

  fflush(stdout);

}

/******************************************************************************
 * Degree Centrality Testing Routines
 *****************************************************************************/
/**
 * @brief A simple CPU-based reference TOPK implementation.
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 */
struct compare_second_only 
{
  template <typename T1, typename T2>
  bool operator()(const std::pair<T1, T2>& p1, const std::pair<T1, T2>& p2)
  {
    return p1.second > p2. second;
  }
};

template<typename VertexId, 
	 typename Value, 
	 typename SizeT>
void SimpleReferenceTopK(const Csr<VertexId, Value, SizeT> &graph_n,
			 const Csr<VertexId, Value, SizeT> &graph_r,
			 VertexId *ref_node_id,
			 Value    *ref_degrees,
			 SizeT    top_nodes)
{
  
  printf("CPU reference test.\n");
  CpuTimer cpu_timer;
  
  // preparation
  Value    *ref_degrees_n = (Value*)malloc(sizeof(Value) * graph_n.nodes);
  Value    *ref_degrees_r = (Value*)malloc(sizeof(Value) * graph_r.nodes);
  std::vector< pair<int, int> > results; 
  
  for (SizeT node = 0; node < graph_n.nodes; ++node)
  {
    ref_degrees_n[node] = graph_n.row_offsets[node+1] - graph_n.row_offsets[node];
    ref_degrees_r[node] = graph_r.row_offsets[node+1] - graph_r.row_offsets[node];
  }
  
  cpu_timer.Start();

  for (SizeT node = 0; node < graph_n.nodes; ++node)
  {
    ref_degrees_n[node] = ref_degrees_n[node] + ref_degrees_r[node];
    results.push_back( std::make_pair (node, ref_degrees_n[node]) );
  }
  
  // pair sort according to second elements - degree centrality
  std::stable_sort(results.begin(), results.end(), compare_second_only());
  
  for (SizeT itr = 0; itr < top_nodes; ++itr)
  {
    ref_node_id[itr] = results[itr].first;
    ref_degrees[itr] = results[itr].second;
  }

  cpu_timer.Stop();
  float elapsed_cpu = cpu_timer.ElapsedMillis();
  printf("==> CPU Degree Centrality finished in %lf msec.\n", elapsed_cpu);
  
  // clean up if neccessary
  if (ref_degrees_n) { free(ref_degrees_n); } 
  if (ref_degrees_r) { free(ref_degrees_r); }
  results.clear();
  
}

/**
 * @brief Run TopK tests
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 * @tparam INSTRUMENT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] max_grid_size Maximum CTA occupancy
 * @param[in] num_gpus Number of GPUs
 *
 */
template <
  typename VertexId,
  typename Value,
  typename SizeT,
  bool INSTRUMENT>
void RunTests(const Csr<VertexId, Value, SizeT> &graph,
	      const Csr<VertexId, Value, SizeT> &graph_inv,
	      CommandLineArgs                   &args,
 	      int                               max_grid_size,
	      int                               num_gpus,
	      int                               top_nodes,
	      CudaContext                       &context)
{
  
  // define the problem data structure for graph primitive
  typedef TOPKProblem<VertexId, SizeT, Value> Problem;
  
  // INSTRUMENT specifies whether we want to keep such statistical data
  // Allocate TopK enactor map 
  TOPKEnactor<INSTRUMENT> topk_enactor(g_verbose);
  
  // allocate problem on GPU
  // create a pointer of the TOPKProblem type 
  Problem *topk_problem = new Problem;
  
  // reset top_nodes if input k > total number of nodes
  if (top_nodes > graph.nodes) 
  { 
    top_nodes = graph.nodes; 
  }
  
  // malloc host memory
  VertexId *h_node_id   = (VertexId*)malloc(sizeof(VertexId) * top_nodes);
  VertexId *ref_node_id = (VertexId*)malloc(sizeof(VertexId) * top_nodes);
  Value    *h_degrees   = (  Value* )malloc(sizeof(  Value ) * top_nodes);
  Value    *ref_degrees = (  Value* )malloc(sizeof(  Value ) * top_nodes);

  // copy data from CPU to GPU
  // initialize data members in DataSlice for graph
  util::GRError(topk_problem->Init(g_stream_from_host,
				   graph,
				   graph_inv,
				   num_gpus), 
		"Problem TOPK Initialization Failed", __FILE__, __LINE__);
  
  // perform degree centrality
  GpuTimer gpu_timer; // Record the kernel running time
  
  // reset values in DataSlice for graph
  util::GRError(topk_problem->Reset(topk_enactor.GetFrontierType()), 
		"TOPK Problem Data Reset Failed", __FILE__, __LINE__);
  
  gpu_timer.Start();
  // launch topk enactor
  util::GRError(topk_enactor.template Enact<Problem>(context, 
						     topk_problem, 
						     top_nodes, 
						     max_grid_size), 
		"TOPK Problem Enact Failed", __FILE__, __LINE__);
  
  gpu_timer.Stop();
  
  float elapsed_gpu = gpu_timer.ElapsedMillis();
  printf("==> GPU TopK Degree Centrality finished in %lf msec.\n", elapsed_gpu);
  
  // copy out results back to CPU from GPU using Extract
  util::GRError(topk_problem->Extract(h_node_id,
				      h_degrees,
				      top_nodes),
		"TOPK Problem Data Extraction Failed", __FILE__, __LINE__);
  
  // display solution
  DisplaySolution(h_node_id, h_degrees, top_nodes);
  
  // validation
  SimpleReferenceTopK(graph, graph_inv, ref_node_id, ref_degrees, top_nodes);
  
  int error_num = CompareResults(h_node_id, ref_node_id, top_nodes, true);
  if (error_num > 0)
  {
    printf("INCOREECT! %d error(s) occured. \n", error_num);
  }
  printf("\n");
  
  // cleanup if neccessary
  if (topk_problem) { delete topk_problem; }
  if (h_node_id)    {   free(h_node_id);   }
  if (h_degrees)    {   free(h_degrees);   }

  hipDeviceSynchronize();
}

/**
 * @brief RunTests entry
 *
 * @tparam VertexId
 * @tparam Value
 * @tparam SizeT
 *
 * @param[in] graph Reference to the CSR graph we process on
 * @param[in] args Reference to the command line arguments
 */
template <typename VertexId,
	  typename Value,
	  typename SizeT>
void RunTests(Csr<VertexId, Value, SizeT> &graph,
	      Csr<VertexId, Value, SizeT> &graph_inv,
	      CommandLineArgs		  &args,
	      SizeT                       top_nodes,
	      CudaContext                 &context)
{
  bool 	instrumented 	= false;
  int 	max_grid_size 	= 0;            
  int 	num_gpus	= 1;            
    
  instrumented = args.CheckCmdLineFlag("instrumented");
    
  g_quick = args.CheckCmdLineFlag("quick");
  g_verbose = args.CheckCmdLineFlag("v");
  
  if (instrumented) 
  {
    RunTests<VertexId, Value, SizeT, true>(graph,
					   graph_inv,
					   args,
					   max_grid_size,
					   num_gpus,
					   top_nodes,
					   context);
  }
  else 
  {
    RunTests<VertexId, Value, SizeT, false>(graph,
					    graph_inv,
					    args,
					    max_grid_size,
					    num_gpus,
					    top_nodes,
					    context);
  }
}

/******************************************************************************
 * Main
 ******************************************************************************/
int main(int argc, char** argv)
{
  CommandLineArgs args(argc, argv);
  
  if ((argc < 2) || (args.CheckCmdLineFlag("help"))) 
  {
    Usage();
    return 1;
  }
  
  //DeviceInit(args);
  //hipSetDeviceFlags(hipDeviceMapHost);
  int dev = 0;
  int top_nodes;

  args.GetCmdLineArgument("device", dev);
  args.GetCmdLineArgument("top", top_nodes);
  
  mgpu::ContextPtr context = mgpu::CreateCudaDevice(dev);
  //srand(0);			// Presently deterministic
  //srand(time(NULL));
  
  // Parse graph-contruction params
  g_undirected = false;
  
  std::string graph_type = argv[1];
  int flags = args.ParsedArgc();
  int graph_args = argc - flags - 1;
  
  if (graph_args < 1) 
  {
    Usage();
    return 1;
  }
  
  //
  // Construct graph and perform
  //
  if (graph_type == "market") 
  {

    // Matrix-market coordinate-formatted graph file
    
    typedef int VertexId;	// Use as the node identifier type
    typedef int Value;	        // Use as the value type
    typedef int SizeT;	        // Use as the graph size type
    
    Csr<VertexId, Value, SizeT> csr(false);
    Csr<VertexId, Value, SizeT> csr_inv(false);
      
    // Default value for stream_from_host is false
    if (graph_args < 1)
    {
      Usage();
      return 1;
    }
      
    char *market_filename = (graph_args == 2) ? argv[2] : NULL;
    
    // BuildMarketGraph() reads a mtx file into CSR data structure
    // Template argumet = true because the graph has edge weights
    // read in non-inversed graph
    if (graphio::BuildMarketGraph<true>(market_filename,
					csr,
					g_undirected,
					false) != 0) // no inverse graph
    { return 1; }

    // read in inversed graph
    if (graphio::BuildMarketGraph<true>(market_filename,
					csr_inv,
					g_undirected,
					true) != 0) // inversed graph
    { return 1; }

    // run gpu tests
    RunTests(csr, csr_inv, args, top_nodes, *context);

  }
  else 
  {
    // unknown graph type
    fprintf(stderr, "Unspecified graph type\n");
    return 1;
  }
  
  return 0;
}

/* end */
