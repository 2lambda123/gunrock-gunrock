// XXX: dummy template for unit testing

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <gunrock/formats/csr.hxx>

void test_csr()
{
  using namespace gunrock;
  using namespace gunrock::format;

  using offset_t = int;
  using index_t = int;
  using value_t = float;

  error::error_t status = hipSuccess;

  // CSR array with default constructor
  csr_t<offset_t, index_t, value_t> csr;

  // CSR array with space allocated (4x4x4)
  std::size_t r, c, nnz = 4;
  memory::memory_space_t location = memory::memory_space_t::host;
  csr_t<offset_t, index_t, value_t> _csr(r, c, nnz, location);

  // CSR array with pre-populated pointers (4x4x4)
  // V         = [ 5 8 3 6 ]
  // COL_INDEX = [ 0 1 2 1 ]
  // ROW_INDEX = [ 0 0 2 3 4 ]
  offset_t *Ap = memory::allocate<offset_t>((r+1)*sizeof(offset_t), location);
  index_t *Aj = memory::allocate<index_t>((nnz)*sizeof(index_t), location);
  value_t *Ax = memory::allocate<value_t>((nnz)*sizeof(value_t), location);

  // XXX: ugly way to initialize these, but it works.
  Ap[0] = 0; Ap[1] = 0; Ap[2] = 2; Ap[3] = 3; Ap[4] = 4;
  Aj[0] = 0; Aj[1] = 1; Aj[2] = 2; Aj[3] = 3;
  Ax[0] = 5; Ax[1] = 8; Ax[2] = 3; Ax[3] = 6;

  csr_t<offset_t, index_t, value_t> __csr(r, c, nnz,
    Ap, Aj, Ax, location);
}

int
main(int argc, char** argv)
{
  test_csr();
  return;
}