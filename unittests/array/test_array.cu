#include "hip/hip_runtime.h"
#include <cstdlib>  // EXIT_SUCCESS

#include <gunrock/container/array.hxx>
#include <gunrock/error.hxx>  // error checking

template <std::size_t N, typename T>
__global__ void kernel(T a) {
  int idx = threadIdx.x + (blockDim.x * blockIdx.x);
  if (idx >= N)
    return;

  a[idx] = (float)idx;
  printf("a[%i] = %f\n", idx, a[idx]);
}

void test_array() {
  using namespace gunrock;

  error::error_t status = hipSuccess;

  constexpr std::size_t N = 10;

  array<float, N> a;

  float* pointer = a.data();
  const float* const_pointer = a.data();

  std::size_t size = a.size();
  std::size_t max_size = a.max_size();
  bool is_empty = a.empty();

  std::cout << "Array.size() = " << size << std::endl;
  std::cout << "Array.max_size() = " << max_size << std::endl;
  std::cout << "Is Array Empty? " << std::boolalpha << is_empty << std::endl;

  status = hipDeviceSynchronize();
  if (hipSuccess != status)
    throw error::exception_t(status);

  kernel<N><<<1, N>>>(a);

  status = hipDeviceSynchronize();
  if (hipSuccess != status)
    throw error::exception_t(status);

  // Segmentation fault; no host support
  // XXX: this is trivial to add using
  // a thrust::host_vector, but we have
  // to handle move symantics ourselves,
  // and that is when things get really
  // complicated. I will consider this if
  // find it useful.
  // a[0] = 0;
}

int main(int argc, char** argv) {
  test_array();
  return EXIT_SUCCESS;
}