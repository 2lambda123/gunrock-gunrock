#include "hip/hip_runtime.h"
#include <gunrock/error.hxx>
#include <gunrock/container/array.cuh>

template<std::size_t N, typename T>
__global__ void kernel(T a) 
{
  int idx = threadIdx.x + (blockDim.x * blockIdx.x);
  if (idx >= N) return;

  a[idx] = (float)idx;
  printf("a[%i] = %f\n", idx, a[idx]);
}

void test_array()
{
  using namespace gunrock;

  error::error_t status = hipSuccess;

  constexpr std::size_t N     = 10;

  array<float, N>               a;

  float* pointer              = a.data();
  const float* const_pointer  = a.data();

  std::size_t size            = a.size();
  std::size_t max_size        = a.max_size();
  bool is_empty               = a.empty();

  status = hipDeviceSynchronize();
  if(hipSuccess != status) throw error::exception_t(status);

  kernel<N><<<1, N>>>(a);
  
  status = hipDeviceSynchronize();
  if(hipSuccess != status) throw error::exception_t(status);

  // Segmentation fault; no host support
  // XXX: this is trivial to add using
  // a thrust::host_vector, but we have
  // to handle move symantics ourselves,
  // and that is when things get really
  // complicated. I will consider this if
  // find it useful.
  // a[0] = 0;
}

int
main(int argc, char** argv)
{
  test_array();
  return;
}