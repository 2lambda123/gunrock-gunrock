#include "hip/hip_runtime.h"
// XXX: dummy template for unit testing

#define THRUST_IGNORE_CUB_VERSION_CHECK

#include <gunrock/container/array.cuh>
typedef hipError_t test_error_t;

template<typename T>
__global__ void kernel(T a) 
{
  int idx = threadIdx.x + (blockDim.x * blockIdx.x);
  if (idx > a.size()) return;

  a[idx] = (float)idx;
  printf("a[%i] = %f\n", idx, a[idx]);
}

test_error_t
test_array()
{
  using namespace gunrock;
  using namespace container::dense;

  test_error_t status         = hipSuccess;
  const std::size_t N         = 10;

  array<float, N>               a;

  float* pointer              = a.data();
  const float* const_pointer  = a.data();

  std::size_t size            = a.size();
  std::size_t max_size        = a.max_size();
  bool is_empty               = a.empty();

  hipDeviceSynchronize();
  kernel<<<1, N>>>(a);
  hipDeviceSynchronize();

  // Segmentation fault; no host support
  // a[0] = 0;

  return status;
}

int
main(int argc, char** argv)
{
  return test_array();
}