// XXX: dummy template for unit testing

#include <gunrock/data_structs/array.cuh>
typedef hipError_t error_t;

error_t
test_array()
{
  using namespace gunrock;

  error_t retval = hipSuccess;
  size_t N = 128;
  gunrock::datastruct::dense::array<int, N> a;

  return retval;
}

int
main(int argc, char** argv)
{
  return test_array();
}