#include <gunrock/error.hxx>
#include <gunrock/formats/formats.hxx>

void test_format()
{
  using namespace gunrock;
  using namespace gunrock::format;

  using offset_t = int;
  using index_t = int;
  using value_t = float;

  error::error_t status = hipSuccess;

  // CSR, CSC, COO classes with default constructors
  csr_t<offset_t, index_t, value_t> csr;
  csc_t<offset_t, index_t, value_t> csc;
  coo_t<index_t, index_t, value_t> coo;
}

int
main(int argc, char** argv)
{
  test_format();
  return;
}