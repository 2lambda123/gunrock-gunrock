#include <gunrock/algorithms/sssp.hxx>
#include "sssp_cpu.hxx"  // Reference implementation
#include <gunrock/util/performance.hxx>
#include <gunrock/io/parameters.hxx>

using namespace gunrock;
using namespace memory;

void test_sssp(int num_arguments, char** argument_array) {
  // --
  // Define types

  using vertex_t = int;
  using edge_t = int;
  using weight_t = float;

  using csr_t =
      format::csr_t<memory_space_t::device, vertex_t, edge_t, weight_t>;

  // --
  // IO

  gunrock::io::cli::parameters_t params(num_arguments, argument_array,
                                        "Single Source Shortest Path");

  csr_t csr;
  io::matrix_market_t<vertex_t, edge_t, weight_t> mm;

  if (params.binary) {
    csr.read_binary(params.filename);
  } else {
    csr.from_coo(mm.load(params.filename));
  }

  // --
  // Build graph

  auto G = graph::build::from_csr<memory_space_t::device, graph::view_t::csr>(
      csr.number_of_rows,               // rows
      csr.number_of_columns,            // columns
      csr.number_of_nonzeros,           // nonzeros
      csr.row_offsets.data().get(),     // row_offsets
      csr.column_indices.data().get(),  // column_indices
      csr.nonzero_values.data().get()   // values
  );  // supports row_indices and column_offsets (default = nullptr)

  // --
  // Params and memory allocation
  srand(time(NULL));

  vertex_t n_vertices = G.get_number_of_vertices();
  vertex_t single_source = 0;  // rand() % n_vertices;
  std::cout << "Single Source = " << single_source << std::endl;

  // --
  // GPU Run

  /// An example of how one can use std::shared_ptr to allocate memory on the
  /// GPU, using a custom deleter that automatically handles deletion of the
  /// memory.
  // std::shared_ptr<weight_t> distances(
  //     allocate<weight_t>(n_vertices * sizeof(weight_t)),
  //     deleter_t<weight_t>());
  // std::shared_ptr<vertex_t> predecessors(
  //     allocate<vertex_t>(n_vertices * sizeof(vertex_t)),
  //     deleter_t<vertex_t>());

  thrust::device_vector<weight_t> distances(n_vertices);
  thrust::device_vector<vertex_t> predecessors(n_vertices);
  thrust::device_vector<int> edges_visited(1);
  thrust::device_vector<int> vertices_visited(1);
  int search_depth = 0;

  std::vector<float> run_times;
  for (int i = 0; i < params.num_runs; i++) {
    run_times.push_back(gunrock::sssp::run(
        G, single_source, params.collect_metrics, distances.data().get(),
        predecessors.data().get(), edges_visited.data().get(),
        vertices_visited.data().get(), &search_depth));
  }

  print::head(distances, 40, "GPU distances");
  std::cout << "GPU Elapsed Time : " << run_times[params.num_runs - 1]
            << " (ms)" << std::endl;

  // --
  // CPU Run

  if (params.validate) {
    thrust::host_vector<weight_t> h_distances(n_vertices);
    thrust::host_vector<vertex_t> h_predecessors(n_vertices);

    float cpu_elapsed = sssp_cpu::run<csr_t, vertex_t, edge_t, weight_t>(
        csr, single_source, h_distances.data(), h_predecessors.data());

    int n_errors =
        util::compare(distances.data().get(), h_distances.data(), n_vertices);

    print::head(h_distances, 40, "CPU Distances");

    std::cout << "CPU Elapsed Time : " << cpu_elapsed << " (ms)" << std::endl;
    std::cout << "Number of errors : " << n_errors << std::endl;
  }

  // --
  // Run performance evaluation

  if (params.collect_metrics) {
    vertex_t n_edges = G.get_number_of_edges();

    // For BFS - the number of nodes visited is just 2 * edges_visited
    thrust::host_vector<int> h_edges_visited = edges_visited;
    thrust::host_vector<int> h_vertices_visited = vertices_visited;
    gunrock::util::stats::get_performance_stats(
        h_edges_visited[0], h_vertices_visited[0], n_edges, n_vertices,
        search_depth, run_times, "sssp", params.filename, "market",
        params.json_dir, params.json_file);
  }
}

int main(int argc, char** argv) {
  test_sssp(argc, argv);
}
