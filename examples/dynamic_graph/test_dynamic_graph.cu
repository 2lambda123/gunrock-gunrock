// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_dynamic_graph.cu
 *
 * @brief Simple test driver program for dynamic graph building.
 */

#include <gunrock/gunrock.h>


// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph defintions
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>

#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
    
    using WeightedGraphT = app::TestGraph<VertexT, SizeT, ValueT,
                                    graph::HAS_EDGE_VALUES | 
                                    graph::HAS_CSR | 
                                    graph::HAS_COO |
                                    graph::HAS_DYN>;

    using CSRGraphT = app::TestGraph<VertexT, SizeT, ValueT,
                                    graph::HAS_EDGE_VALUES | 
                                    graph::HAS_CSR>;

    hipError_t retval = hipSuccess;
    util::CpuTimer cpu_timer;
    
    WeightedGraphT weighted_graph;

    cpu_timer.Start();
    GUARD_CU(graphio::LoadGraph(parameters, weighted_graph));
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    return retval;
  }
};

int main(int argc, char **argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test sssp");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  return app::Switch_Types<app::VERTEXT_U32B |
                           app::SIZET_U32B | 
                           app::VALUET_S32B | app::DIRECTED | app::UNDIRECTED>(
      parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
