// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_dynamic_graph.cu
 *
 * @brief Simple test driver program for dynamic graph building.
 */

#include <gunrock/gunrock.h>


// Utilities and correctness-checking
#include <gunrock/util/test_utils.cuh>

// Graph defintions
#include <gunrock/graphio/graphio.cuh>
#include <gunrock/app/app_base.cuh>

#include <gunrock/app/test_base.cuh>

using namespace gunrock;

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters &parameters, VertexT v, SizeT s, ValueT val) {
    typedef typename app::TestGraph<VertexT, SizeT, ValueT,
                                    graph::HAS_EDGE_VALUES | graph::HAS_CSR | graph::HAS_DYN>
        GraphT;
    typedef typename GraphT::CsrT CsrT;
    typedef typename GraphT::DynT DynT;

    hipError_t retval = hipSuccess;
    util::CpuTimer cpu_timer;
    GraphT graph;  // graph we process on

    cpu_timer.Start();
    GUARD_CU(graphio::LoadGraph(parameters, graph));
    // force edge values to be 1, don't enable this unless you really want to
    // for (SizeT e=0; e < graph.edges; e++)
    //    graph.CsrT::edge_values[e] = 1;
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());
    // GUARD_CU(graph.CsrT::edge_values.Print("", 100));
    // util::PrintMsg("sizeof(VertexT) = " + std::to_string(sizeof(VertexT))
    //    + ", sizeof(SizeT) = " + std::to_string(sizeof(SizeT))
    //    + ", sizeof(ValueT) = " + std::to_string(sizeof(ValueT)));


    return retval;
  }
};

int main(int argc, char **argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test sssp");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  return app::Switch_Types<app::VERTEXT_U32B |
                           app::SIZET_U32B | 
                           app::VALUET_S32B | app::DIRECTED | app::UNDIRECTED>(
      parameters, main_struct());


  //return app::Switch_Types<app::VERTEXT_U32B | app::VERTEXT_U64B |
  //                         app::SIZET_U32B | app::SIZET_U64B |
  //                         app::VALUET_S32B | app::DIRECTED | app::UNDIRECTED>(
  //    parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
