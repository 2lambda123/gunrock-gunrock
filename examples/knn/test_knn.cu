// ----------------------------------------------------------------
// Gunrock -- Fast and Efficient GPU Graph Library
// ----------------------------------------------------------------
// This source code is distributed under the terms of LICENSE.TXT
// in the root directory of this source distribution.
// ----------------------------------------------------------------

/**
 * @file
 * test_knn.cu
 *
 * @brief Simple test driver program for Gunrock template.
 */

// KNN includes
#include <gunrock/app/knn/knn_app.cu>
#include <gunrock/app/knn/knn_helpers.cuh>

// App and test base includes
#include <gunrock/app/test_base.cuh>

//#define KNN_DEBUG 1
#ifdef KNN_DEBUG
    #define debug(a...) fprintf(stderr, a)
#else
    #define debug(a...)
#endif

using namespace gunrock;

namespace APP_NAMESPACE = app::knn;

/**
 * @brief Compute euclidean distance
 * @param dim Number of dimensions (2D, 3D ... ND)
 * @param points Points array to get the x, y, z...
 * @param p1 and p2 points to be compared
 * info \return distance value
 */
 template<typename SizeT, typename ValueT>
 __device__ __host__
 ValueT distances(const SizeT dim, ValueT* points, SizeT p1, SizeT p2) {
     // Get dimensional of labels
     ValueT result = (ValueT) 0;
     // p1 = (x_1, x_2, ..., x_dim)
     // p2 = (y_1, y_2, ..., y_dim)
     for (int i=0; i<dim; ++i){
         //(x_i - y_i)^2
         ValueT diff = points[p1 * dim + i] - points[p2 * dim + i];
         result += diff*diff;
     }
     return result;
 }

/******************************************************************************
 * Main
 ******************************************************************************/

/**
 * @brief Enclosure to the main function
 */
struct main_struct {
  /**
   * @brief the actual main function, after type switching
   * @tparam VertexT    Type of vertex identifier
   * @tparam SizeT      Type of graph size, i.e. type of edge identifier
   * @tparam ValueT     Type of edge values
   * @param  parameters Command line parameters
   * @param  v,s,val    Place holders for type deduction
   * \return hipError_t error message(s), if any
   */
  template <typename VertexT,  // Use int as the vertex identifier
            typename SizeT,    // Use int as the graph size type
            typename ValueT>   // Use int as the value type
  hipError_t
  operator()(util::Parameters& parameters, VertexT v, SizeT s, ValueT val) {
    // CLI parameters
    bool quick = parameters.Get<bool>("quick");
    bool quiet = parameters.Get<bool>("quiet");

    // Get n dimension tuplets
    std::string labels_file = parameters.Get<std::string>("labels-file");
    util::PrintMsg("Points File Input: " + labels_file, !quiet);

    typedef typename app::TestGraph<VertexT, SizeT, ValueT, graph::HAS_CSR>
        GraphT;
    // Creating empty graph
    GraphT graph;

    hipError_t retval = hipSuccess;

    // Initialization of the points array
    util::Array1D<SizeT, ValueT> points;
    //Initialization is moved to gunrock::graphio::labels::Read ... ReadLabelsStream
    //GUARD_CU(points.Allocate(n*dim, util::HOST));
    
    util::CpuTimer cpu_timer;
    cpu_timer.Start();
    // graphio::labels is setting "n" and "dim"
    retval = gunrock::graphio::labels::Read(parameters, points);
    if (retval){
        util::PrintMsg("Reading error\n");
        return retval;
    }
    cpu_timer.Stop();
    parameters.Set("load-time", cpu_timer.ElapsedMillis());

    // Get number of points
    SizeT n = parameters.Get<SizeT>("n");
   
    // Get dimensional of space
    SizeT dim = parameters.Get<SizeT>("dim");

    // Get number of nearest neighbors, default k = 10
    SizeT k = parameters.Get<SizeT>("k");

    if (k >= n){
        util::PrintMsg("k has to be at most n-1", !quiet);
        return retval;
    }
 
#ifdef KNN_DEBUG
    // Debug of points:
    debug("debug points\n");
    for (int i=0; i<n; ++i){
        debug("for point %d: ", i);
        for (int j=0; j<dim; ++j){
            debug("%.lf ", points[i*dim + j]);
        }
        debug("\n");
    }
#endif

    // Reference result on CPU
    SizeT* ref_knns = NULL;
    SizeT* h_knns = (SizeT*)malloc(sizeof(SizeT) * n * k);

    if (!quick) {
      // Init datastructures for reference result on GPU
      ref_knns = (SizeT*)malloc(sizeof(SizeT) * n * k);

      // If not in `quick` mode, compute CPU reference implementation
      util::PrintMsg("__________________________", !quiet);
      util::PrintMsg("______ CPU Reference _____", !quiet);

      float elapsed = app::knn::CPU_Reference<VertexT, SizeT, ValueT>(
              points, n, dim, k, ref_knns, quiet);

      util::PrintMsg("--------------------------\n Elapsed: " + 
              std::to_string(elapsed), !quiet);
      util::PrintMsg("__________________________", !quiet);
      parameters.Set("cpu-elapsed", elapsed);
    }

    std::vector<std::string> switches{"advance-mode"};

    GUARD_CU((app::Switch_Parameters(parameters, graph, switches,
        [n, dim, k, h_knns, points, ref_knns]
        (util::Parameters& parameters, GraphT& graph) {
            return app::knn::RunTests(parameters, points, graph, n, dim, k, 
                    h_knns, ref_knns, util::DEVICE);
        })));

    if (!quick) {
      delete[] ref_knns;
    }

    return retval;
  }
};

int main(int argc, char** argv) {
  hipError_t retval = hipSuccess;
  util::Parameters parameters("test knn");
  GUARD_CU(graphio::UseParameters(parameters));
  GUARD_CU(app::knn::UseParameters(parameters));
  GUARD_CU(app::UseParameters_test(parameters));
  GUARD_CU(parameters.Parse_CommandLine(argc, argv));
  if (parameters.Get<bool>("help")) {
    parameters.Print_Help();
    return hipSuccess;
  }
  GUARD_CU(parameters.Check_Required());

  app::Switch_Types<app::VERTEXT_U32B | app::VERTEXT_U64B |
                           app::SIZET_U32B | app::SIZET_U64B |
                           app::VALUET_F32B | app::UNDIRECTED>(
      parameters, main_struct());
}

// Leave this at the end of the file
// Local Variables:
// mode:c++
// c-file-style: "NVIDIA"
// End:
